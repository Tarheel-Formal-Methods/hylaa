#include "hip/hip_runtime.h"
// Dung Tran & Stanley Bak
// Krylov subspace - based simulation using Gpu- Cusp / Cuda for sparse ode
// June 2017

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include <cusp/array1d.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/multiply.h>
#include <cusp/multiply.h>
#include <cusp/print.h>

// CUDA runtime
//#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "gpu_util.h"

typedef double FLOAT_TYPE;

// anyArray is only used to identify if CPU (do nothing) or GPU (initialize handle)
void createCublasHandle(hipblasHandle_t &cublasHandle,
                        cusp::array1d<FLOAT_TYPE, cusp::device_memory> *anyArray)
{
    if (cublasHandle != 0)
        error("cublasHandle initialized twice");

    if (hipblasCreate(&cublasHandle) != HIPBLAS_STATUS_SUCCESS)
        error("hipblasCreate() failed");

    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
}

// anyArray is only used to identify if CPU (do nothing) or GPU (initialize handle)
void createCublasHandle(hipblasHandle_t &cublasHandle,
                        cusp::array1d<FLOAT_TYPE, cusp::host_memory> *anyArray)
{
    if (cublasHandle != 0)
        error("cublasHandle initialized twice");
}

void myDot(hipblasHandle_t &cublasHandle, unsigned long size,
           cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &a,
           cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &b,
           cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &resultView, int resultIndex)
{
    // cpu implementation
    FLOAT_TYPE d = cusp::blas::dot(a, b);

    resultView[resultIndex] = d;
}

void myDot(hipblasHandle_t &cublasHandle, unsigned long size,
           cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &a,
           cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &b,
           cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &resultView, int resultIndex)
{
    double *x = thrust::raw_pointer_cast(&a[0]);
    double *y = thrust::raw_pointer_cast(&b[0]);
    double *result = thrust::raw_pointer_cast(&resultView[resultIndex]);

    if (hipblasDdot(cublasHandle, size, x, 1, y, 1, result) != HIPBLAS_STATUS_SUCCESS)
        error("hipblasDdot() failed");
}

// subtract dots * prevVec from curVec using axpy
void myAxpy(hipblasHandle_t &cublasHandle,
            // numsView: [-1, 0, 1, temp storage]
            cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &numsView,
            cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &a,
            cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &resView,
            cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &hView, int hIndex)
{
    // cpu implementation
    cusp::blas::axpy(a, resView, -hView[hIndex]);
}

// subtract dots * prevVec from curVec using axpy
void myAxpy(hipblasHandle_t &cublasHandle,
            // numsView: [-1, 0, 1, temp storage]
            cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &numsView,
            cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &a,
            cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &resView,
            cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &hView, int hIndex)
{
    // gpu implementation
    // cusp::blas::axpy(a, b, -hView[hIndex]);

    int count = a.size();

    double *minusOne = thrust::raw_pointer_cast(&numsView[0]);
    double *zero = thrust::raw_pointer_cast(&numsView[1]);
    double *temp = thrust::raw_pointer_cast(&numsView[3]);
    double *x = thrust::raw_pointer_cast(&a[0]);
    double *res = thrust::raw_pointer_cast(&resView[0]);
    double *h = thrust::raw_pointer_cast(&hView[hIndex]);

    // copy h to temp
    if (hipblasDcopy(cublasHandle, 1, h, 1, temp, 1) != HIPBLAS_STATUS_SUCCESS)
        error("hipblasDcopy() failed");

    // scale temp by -1
    if (hipblasDscal(cublasHandle, 1, minusOne, temp, 1) != HIPBLAS_STATUS_SUCCESS)
        error("hipblasDscal() failed");

    // do the axpy (alpha = temp)
    if (hipblasDaxpy(cublasHandle, count, temp, x, 1, res, 1) != HIPBLAS_STATUS_SUCCESS)
        error("hipblasDaxpy() failed");
}

template <class MEMORY_TYPE>
class CuspData
{
    typedef cusp::array1d<FLOAT_TYPE, MEMORY_TYPE> Array1d;
    typedef typename Array1d::view Array1dView;

    typedef cusp::array2d<FLOAT_TYPE, MEMORY_TYPE> Array2d;
    typedef typename Array2d::view Array2dView;

    typedef cusp::array1d<FLOAT_TYPE, cusp::host_memory> HostFloatArray1d;
    typedef typename HostFloatArray1d::view HostFloatArray1dView;

    typedef cusp::array1d<int, cusp::host_memory> HostIntArray1d;
    typedef typename HostIntArray1d::view HostIntArray1dView;

    typedef cusp::coo_matrix<int, FLOAT_TYPE, MEMORY_TYPE> CooMatrix;

    typedef cusp::hyb_matrix<int, FLOAT_TYPE, MEMORY_TYPE> HybMatrix;

    typedef cusp::csr_matrix<int, FLOAT_TYPE, cusp::host_memory> HostCsrMatrix;
    typedef typename HostCsrMatrix::view HostCsrMatrixView;

   private:
    HybMatrix *aMatrix;
    HybMatrix *keyDirMatrix;

    Array1d *vMatrix;     // (i+1) * n
    Array1d *hMatrix;     // (i+1) * i
    Array1d *vProjected;  // k * (i+1)

    unsigned long _n;  // number of dimensions in the system
    unsigned long _k;  // number of key directions
    unsigned long _i;  // number of arnoldi iterations

    // profiling variables
    bool useProfiling;
    unsigned long aMatrixNonzeros;
    unsigned long keyDirMatrixNonzeros;

    // cublas variables
    hipblasHandle_t cublasHandle;
    Array1d *cuspNums;  // [-1, 0, 1, temp_val]

   public:
    GpuUtil util;  // timers and other utility functions

    CuspData(bool useCpu) : util(useCpu), cublasHandle(0)
    {
        aMatrix = 0;
        keyDirMatrix = 0;

        vMatrix = 0;
        hMatrix = 0;
        vProjected = 0;

        cuspNums = 0;

        reset();  // this resets all variables
    }

    ~CuspData()
    {
        reset();

        hipDeviceReset();
    }

    void reset()
    {
        if (aMatrix != 0)
        {
            delete aMatrix;
            aMatrix = 0;
        }

        if (keyDirMatrix != 0)
        {
            delete keyDirMatrix;
            keyDirMatrix = 0;
        }

        if (vMatrix != 0)
        {
            delete vMatrix;
            vMatrix = 0;
        }

        if (hMatrix != 0)
        {
            delete hMatrix;
            hMatrix = 0;
        }

        if (vProjected != 0)
        {
            delete vProjected;
            vProjected = 0;
        }

        if (cuspNums != 0)
        {
            delete cuspNums;
            cuspNums = 0;
        }

        if (cublasHandle)
        {
            if (hipblasDestroy(cublasHandle) != HIPBLAS_STATUS_SUCCESS)
                error("hipblasDestroy failed");

            cublasHandle = 0;
        }

        util.clearTimers();

        _n = 0;
        _k = 0;
        _i = 0;

        setUseProfiling(false);
        aMatrixNonzeros = 0;
        keyDirMatrixNonzeros = 0;
    }

    void setUseProfiling(bool enabled)
    {
        useProfiling = enabled;
        util.setUseProfiling(enabled);
    }

    // load A matrix, passed in as a csr matrix
    void loadAMatrix(unsigned long w, unsigned long h, int *rowOffsets, unsigned long rowOffsetsLen,
                     int *colInds, unsigned long colIndsLen, FLOAT_TYPE *values,
                     unsigned long valuesLen)
    {
        if (_n == 0)
            error("loadKeyDirMatrix() called before preallocate() (_n==0)\n");

        if (w != h)
            error("loadAMatrix() expected square A matrix, got w=%lu, h=%lu", w, h);

        if (w != _n)
            error(
                "in loadAMatrix() size (%lu) not to equal dims from preallocate() "
                "(%lu)",
                w, _n);

        if (useProfiling)
        {
            FLOAT_TYPE size = rowOffsetsLen * sizeof(int);
            size += colIndsLen * sizeof(int);
            size += valuesLen * sizeof(FLOAT_TYPE);

            printf(
                "loadAMatrix() with sparse matrix size: %.2f MB (%lu nonzeros). "
                "Memory on device: "
                "%.2f MB\n",
                size / 1024.0 / 1024.0, valuesLen, getFreeMemoryMb());
        }

        aMatrixNonzeros = valuesLen;

        HostIntArray1dView rowOffsetsView(rowOffsets, rowOffsets + rowOffsetsLen);
        HostIntArray1dView colIndsView(colInds, colInds + colIndsLen);
        HostFloatArray1dView valuesView(values, values + colIndsLen);

        HostCsrMatrixView view(_n, _n, valuesLen, rowOffsetsView, colIndsView, valuesView);

        if (aMatrix != 0)
        {
            delete aMatrix;
            aMatrix = 0;
        }

        try
        {
            aMatrix = new HybMatrix(view);
        }
        catch (std::bad_alloc)
        {
            error("memory allocation of aMatrix failed\n");
        }

        // one-time preallocate cusp_nums (shouldn't fail)
        if (cuspNums == 0)
        {
            try
            {
                HostFloatArray1d temp(4);
                temp[0] = -1;
                temp[1] = 0;
                temp[2] = 1;
                temp[3] = 0;
                cuspNums = new Array1d(temp);
            }
            catch (std::bad_alloc)
            {
                error("cuspNums memory allocation failed\n");
            }
        }

        // one-time allocation of cublasHandle
        if (cublasHandle == 0)
            createCublasHandle(cublasHandle, vMatrix);
    }

    // load key dir matrix, passed in as a csr matrix
    void loadKeyDirMatrix(unsigned long w, unsigned long h, int *rowOffsets,
                          unsigned long rowOffsetsLen, int *colInds, unsigned long colIndsLen,
                          FLOAT_TYPE *values, unsigned long valuesLen)
    {
        if (_n == 0)
            error("loadKeyDirMatrix() called before preallocate() (_n==0)\n");

        if (w != _n)
            error(
                "in loadKeyDirMatrix() width (%lu) to equal dims from "
                "preallocate() (%lu)",
                w, _n);

        if (w != _n)
            error(
                "in loadKeyDirMatrix() height (%lu) to equal keyDirMatSize from "
                "preallocate() "
                "(%lu)",
                h, _k);

        if (useProfiling)
        {
            FLOAT_TYPE size = rowOffsetsLen * sizeof(int);
            size += colIndsLen * sizeof(int);
            size += valuesLen * sizeof(FLOAT_TYPE);

            printf(
                "loadKeyDirMatrix() with dense matrix size: %.2f MB, Memory on "
                "device: %.2f MB\n",
                size / 1024.0 / 1024.0, getFreeMemoryMb());
        }

        keyDirMatrixNonzeros = valuesLen;

        HostIntArray1dView rowOffsetsView(rowOffsets, rowOffsets + rowOffsetsLen);
        HostIntArray1dView colIndsView(colInds, colInds + colIndsLen);
        HostFloatArray1dView valuesView(values, values + colIndsLen);

        HostCsrMatrixView view(h, w, valuesLen, rowOffsetsView, colIndsView, valuesView);

        if (keyDirMatrix != 0)
        {
            delete keyDirMatrix;
            keyDirMatrix = 0;
        }

        try
        {
            keyDirMatrix = new HybMatrix(view);
        }
        catch (std::bad_alloc)
        {
            error("memory allocation of keyDirMatrix() failed\n");
        }
    }

    FLOAT_TYPE getFreeMemoryMb()
    {
        unsigned long bytes = util.getFreeMemory();

        return bytes / 1024.0 / 1024.0;
    }

    // frees memory if it was previously allocated, returns false if memory error
    // occurs
    bool preallocateMemory(unsigned long arnoldiIt, unsigned long dims, unsigned long keyDirMatSize)
    {
        bool success = true;

        if (_n != 0 && dims != _n && aMatrix != 0)
        {
            // num dimensions changed... free aMatrix
            delete aMatrix;
            aMatrix = 0;
        }

        if (_k != 0 && keyDirMatSize != _k && keyDirMatrix != 0)
        {
            // num key directions changed
            delete keyDirMatrix;
            keyDirMatrix = 0;
        }

        _n = dims;
        _i = arnoldiIt;
        _k = keyDirMatSize;

        if (useProfiling)
            printf(
                "preallocateMemory() called with profiling, Free memory on "
                "device: %.2f MB\n",
                getFreeMemoryMb());

        // free memory which we will newly allocate

        if (hMatrix != 0)
        {
            delete hMatrix;
            hMatrix = 0;
        }

        if (vMatrix != 0)
        {
            delete vMatrix;
            vMatrix = 0;
        }

        if (vProjected != 0)
        {
            delete vProjected;
            vProjected = 0;
        }

        try
        {
            // preallocate hMatrix, numParInit * iterations * iterations
            unsigned long hMatrixSize = _i * (_i + 1);

            if (useProfiling)
                printf(
                    "Trying to allocate %.2f MB for hMatrix (remaining memory %.2f "
                    "MB)...\n",
                    sizeof(FLOAT_TYPE) * hMatrixSize / 1024.0 / 1024.0, getFreeMemoryMb());

            hMatrix = new Array1d(hMatrixSize, 0);

            // preallocate vMatrix, width = dims * iterations, height = numParInit
            unsigned long vMatrixSize = _n * (_i + 1);

            if (useProfiling)
                printf(
                    "Trying to allocate %.2f MB for vMatrix (remaining memory %.2f "
                    "MB)...\n",
                    sizeof(FLOAT_TYPE) * vMatrixSize / 1024.0 / 1024.0, getFreeMemoryMb());

            vMatrix = new Array1d(vMatrixSize, 0);

            // preallocate vProjected
            unsigned long vProjectedSize = _k * (_i + 1);

            if (useProfiling)
                printf(
                    "Trying to allocate %.2f MB for vProjected (remaining memory %.2f "
                    "MB)...\n",
                    sizeof(FLOAT_TYPE) * vProjectedSize / 1024.0 / 1024.0, getFreeMemoryMb());

            vProjected = new Array1d(vProjectedSize, 0);
        }
        catch (std::bad_alloc)
        {
            if (useProfiling)
                printf("memory allocation failed\n");

            _i = 0;
            _n = 0;
            _k = 0;
            success = false;
        }

        return success;
    }

    // reads/writes from/to vMatrix, writes to hMatrix
    void arnoldi(unsigned long iterations)
    {
        Array1dView cuspNumsView = cuspNums->subarray(0, 4);

        for (unsigned long it = 1; it <= iterations; it++)
        {
            util.tic("sparse matrix vector multiply");
            unsigned long prevRowOffset = _n * (it - 1);
            unsigned long curRowOffset = _n * it;

            Array1dView vecView = vMatrix->subarray(prevRowOffset, _n);
            Array1dView resultView = vMatrix->subarray(curRowOffset, _n);

            cusp::multiply(*aMatrix, vecView, resultView);
            util.toc("sparse matrix vector multiply", 2 * aMatrixNonzeros);

            ////////

            util.tic("dots & axpy");
            unsigned long rowOffset = (it - 1) * (_i + 1);
            resultView = hMatrix->subarray(rowOffset, it);

            rowOffset = _n * it;

            Array1dView curVec = vMatrix->subarray(rowOffset, _n);

            // combined dot/axpy to have modified gram-schmidt orthogonalization
            // (more stable)
            for (unsigned long row = 0; row < it; ++row)
            {
                Array1dView curRow = vMatrix->subarray(row * _n, _n);

                // util.tic("dot");
                myDot(cublasHandle, _n, curVec, curRow, resultView, row);
                // util.toc("dot", 2 * _n);

                rowOffset = _n * row;
                Array1dView prevVec = vMatrix->subarray(rowOffset, _n);

                // util.tic("axpy");
                myAxpy(cublasHandle, cuspNumsView, prevVec, curVec, resultView, row);
                // util.toc("axpy", 2 * _n);
            }

            util.toc("dots & axpy", 2 * 2 * _n * it);

            ////////

            rowOffset = _n * it;
            curVec = vMatrix->subarray(rowOffset, _n);

            util.tic("nrm2");
            FLOAT_TYPE magnitude = cusp::blas::nrm2(curVec);
            util.toc("nrm2");

            // store magnitude in H
            rowOffset = (it - 1) * (_i + 1);
            (*hMatrix)[rowOffset + it] = magnitude;

            // scale vector
            if (magnitude < 1e-10)
            {
                // cusp::blas::scal(curVec, 0.0);

                printf("Break! Profile if this actually helps.\n");
                break;
            }
            else
            {
                util.tic("scale");
                cusp::blas::scal(curVec, 1.0 / magnitude);
                util.toc("scale");
            }
        }
    }

    void projectV(unsigned long iterations)
    {
        // use vMatrix and keyDirMatrix to produce vProjected

        for (unsigned long iteration = 0; iteration <= iterations; ++iteration)
        {
            // Can we do this with a single matrix-matrix mult???
            unsigned long rowOffset = _n * (iteration);
            Array1dView vecView = vMatrix->subarray(rowOffset, _n);

            // result view is in vProjected
            rowOffset = _k * iteration;

            Array1dView resultView = vProjected->subarray(rowOffset, iterations);

            util.tic("project-v sparse matrix vector multiply");
            cusp::multiply(*keyDirMatrix, vecView, resultView);
            util.toc("project-v sparse matrix vector multiply", 2 * keyDirMatrixNonzeros);
        }
    }

    void printProfilingData()
    {
        util.printTimers();
        util.clearTimers();
    }

    // initialize with a passed-in vector
    void initVec(FLOAT_TYPE *vec, unsigned long len)
    {
        util.toc("init arnoldi");

        if (len != _n)
            error("initArnoldi called with bad len = %lu (_n = %lu)\n", len, _n);

        // initialize with zeros
        cusp::blas::fill(*vMatrix, 0.0);
        cusp::blas::fill(*hMatrix, 0.0);
        cusp::blas::fill(*vProjected, 0.0);

        HostFloatArray1dView vecView(vec, vec + len);
        Array1dView vMatrixView = vMatrix->subarray(0, _n);
        cusp::blas::copy(vecView, vMatrixView);

        // sanity check that norm of vec is 1
        FLOAT_TYPE magnitude = cusp::blas::nrm2(vMatrixView);
        FLOAT_TYPE tol = 1e-6;

        if (magnitude < 1.0 - tol || magnitude > 1.0 + tol)
            error("initial arnoldi vec must be normalized first (magnitude was %f)", magnitude);

        util.tic("init arnoldi");
    }

    // initialize with a unit-vector in the given dimention
    void initUnit(unsigned long dim)
    {
        util.tic("init arnoldi");

        if (dim > _n)
            error("initParallelArnoldiV called with single dim=%lu, but _n=%lu", dim, _n);

        // initialize with zeros
        cusp::blas::fill(*vMatrix, 0.0);
        cusp::blas::fill(*hMatrix, 0.0);
        cusp::blas::fill(*vProjected, 0.0);

        // put a 1.0 in the correct spot
        (*vMatrix)[dim] = 1.0;

        util.toc("init arnoldi");
    }

    // copy memory, run arnoldi, project results, copy memory back (everything but init)
    void compute(FLOAT_TYPE *resultH, unsigned long sizeResultH, FLOAT_TYPE *resultPV,
                 unsigned long sizeResultPV)
    {
        if (aMatrix == 0)
            error("arnoldiParallel() called before loadAMatrix()\n");

        if (keyDirMatrix == 0)
            error("arnoldiParallel() called before loadKeyDirMatrix()\n");

        if (_i == 0)
            error("arnoldiParallel() called before preallocate() (_i==0)\n");

        // check expected results sizes
        unsigned long expectedH = _i * (_i + 1);
        unsigned long expectedPV = (_i + 1) * _k;

        if (sizeResultH != expectedH)
            error("Wrong size for resultH with i = %lu. Got %lu, expected %lu.", _i, sizeResultH,
                  expectedH);

        if (sizeResultPV != expectedPV)
            error(
                "Wrong size for resultPV with (i, k) = (%lu, %lu, %lu). Got "
                "%lu, expected %lu.",
                _i, _k, sizeResultPV, expectedPV);

        util.tic("compute");

        util.tic("arnoldi()");
        arnoldi(_i);
        util.toc("arnoldi()");

        // project v_matrix onto keyDirMatrix
        util.tic("projectV()");
        projectV(_i);
        util.toc("projectV()");

        // copying H matrix to np.ndarray
        util.tic("copying H matrix to np.ndarray");
        HostFloatArray1dView hostHView(resultH, resultH + expectedH);
        cusp::blas::copy(*hMatrix, hostHView);  // hostHView = *hMatrix
        util.toc("copying H matrix to np.ndarray");

        // copy vProjected to np.ndarray

        util.tic("copying V-projected matrix to np.ndarray");
        HostFloatArray1dView hostPVView(resultPV, resultPV + expectedPV);
        cusp::blas::copy(*vProjected, hostPVView);  // hostPVView = *vProjected
        util.toc("copying V-projected matrix to np.ndarray");

        util.toc("compute");
    }
};

CuspData<cusp::host_memory> cuspDataCpu(true);
CuspData<cusp::device_memory> cuspDataGpu(false);

extern "C" {
unsigned long hasGpu()
{
    return cuspDataGpu.util.hasGpu();
}

void reset()
{
    cuspDataCpu.reset();
    cuspDataGpu.reset();
}

////// CPU Version
void setUseProfilingCpu(unsigned long enabled)
{
    cuspDataCpu.setUseProfiling(enabled != 0);
}

// as csr matrix
void loadAMatrixCpu(unsigned long w, unsigned long h, int *rowOffsets, unsigned long rowOffsetsLen,
                    int *colInds, unsigned long colIndsLen, FLOAT_TYPE *values,
                    unsigned long valuesLen)
{
    cuspDataCpu.loadAMatrix(w, h, rowOffsets, rowOffsetsLen, colInds, colIndsLen, values,
                            valuesLen);
}

void loadKeyDirMatrixCpu(unsigned long w, unsigned long h, int *rowOffsets,
                         unsigned long rowOffsetsLen, int *colInds, unsigned long colIndsLen,
                         FLOAT_TYPE *values, unsigned long valuesLen)
{
    cuspDataCpu.loadKeyDirMatrix(w, h, rowOffsets, rowOffsetsLen, colInds, colIndsLen, values,
                                 valuesLen);
}

FLOAT_TYPE getFreeMemoryMbCpu()
{
    return cuspDataCpu.getFreeMemoryMb();
}

unsigned long preallocateMemoryCpu(unsigned long arnoldiIt, unsigned long dims,
                                   unsigned long keyDirMatSize)
{
    return cuspDataCpu.preallocateMemory(arnoldiIt, dims, keyDirMatSize) ? 1 : 0;
}

void arnoldiUnitCpu(unsigned long dim, FLOAT_TYPE *resultH, unsigned long sizeResultH,
                    FLOAT_TYPE *resultPV, unsigned long sizeResultPV)
{
    cuspDataCpu.initUnit(dim);
    cuspDataCpu.compute(resultH, sizeResultH, resultPV, sizeResultPV);
}

void arnoldiVecCpu(FLOAT_TYPE *vec, unsigned long vecLen, FLOAT_TYPE *resultH,
                   unsigned long sizeResultH, FLOAT_TYPE *resultPV, unsigned long sizeResultPV)
{
    cuspDataCpu.initVec(vec, vecLen);
    cuspDataCpu.compute(resultH, sizeResultH, resultPV, sizeResultPV);
}

void printProfilingDataCpu()
{
    cuspDataCpu.printProfilingData();
}

////// GPU Version
void setUseProfilingGpu(unsigned long enabled)
{
    cuspDataGpu.setUseProfiling(enabled != 0);
}

// as csr matrix
void loadAMatrixGpu(unsigned long w, unsigned long h, int *rowOffsets, unsigned long rowOffsetsLen,
                    int *colInds, unsigned long colIndsLen, FLOAT_TYPE *values,
                    unsigned long valuesLen)
{
    cuspDataGpu.loadAMatrix(w, h, rowOffsets, rowOffsetsLen, colInds, colIndsLen, values,
                            valuesLen);
}

// as csr matrix
void loadKeyDirMatrixGpu(unsigned long w, unsigned long h, int *rowOffsets,
                         unsigned long rowOffsetsLen, int *colInds, unsigned long colIndsLen,
                         FLOAT_TYPE *values, unsigned long valuesLen)
{
    cuspDataGpu.loadKeyDirMatrix(w, h, rowOffsets, rowOffsetsLen, colInds, colIndsLen, values,
                                 valuesLen);
}

FLOAT_TYPE getFreeMemoryMbGpu()
{
    return cuspDataGpu.getFreeMemoryMb();
}

unsigned long preallocateMemoryGpu(unsigned long arnoldiIterations, unsigned long dims,
                                   unsigned long keyDirMatSize)
{
    return cuspDataGpu.preallocateMemory(arnoldiIterations, dims, keyDirMatSize) ? 1 : 0;
}

void arnoldiUnitGpu(unsigned long dim, FLOAT_TYPE *resultH, unsigned long sizeResultH,
                    FLOAT_TYPE *resultPV, unsigned long sizeResultPV)
{
    cuspDataGpu.initUnit(dim);
    cuspDataGpu.compute(resultH, sizeResultH, resultPV, sizeResultPV);
}

void arnoldiVecGpu(FLOAT_TYPE *vec, unsigned long vecLen, FLOAT_TYPE *resultH,
                   unsigned long sizeResultH, FLOAT_TYPE *resultPV, unsigned long sizeResultPV)
{
    cuspDataGpu.initVec(vec, vecLen);
    cuspDataGpu.compute(resultH, sizeResultH, resultPV, sizeResultPV);
}

void printProfilingDataGpu()
{
    cuspDataGpu.printProfilingData();
}

}  // end extern "C"
