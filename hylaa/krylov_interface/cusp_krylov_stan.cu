// Dung Tran & Stanley Bak
// Krylov subspace - based simulation using Gpu- Cusp / Cuda for sparse ode
// June 2017

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include <cusp/array1d.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/multiply.h>
#include <cusp/multiply.h>
#include <cusp/print.h>

// CUDA runtime
//#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "gpu_util.h"

typedef double FLOAT_TYPE;

void dot_product(hipblasHandle_t &cublasHandle, unsigned long size,
                 cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &a,
                 cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &b,
                 cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &resultView,
                 int resultIndex) {
  // cpu implementation
  FLOAT_TYPE d = cusp::blas::dot(a, b);

  resultView[resultIndex] = d;
}

void dot_product(
    hipblasHandle_t &cublasHandle, unsigned long size,
    cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &a,
    cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &b,
    cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &resultView,
    int resultIndex) {
  // gpu implementation
  if (cublasHandle == 0) {
    if (hipblasCreate(&cublasHandle) != HIPBLAS_STATUS_SUCCESS)
      error("hipblasCreate() failed");

    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
  }

  double *x = thrust::raw_pointer_cast(&a[0]);
  double *y = thrust::raw_pointer_cast(&b[0]);
  double *result = thrust::raw_pointer_cast(&resultView[resultIndex]);

  if (hipblasDdot(cublasHandle, size, x, 1, y, 1, result) !=
      HIPBLAS_STATUS_SUCCESS)
    error("hipblasDdot() failed");
}

// subtract dots * prevVec from curVec using axpy
void do_axpy(hipblasHandle_t &cublasHandle,
             // numsView: [-1, 0, 1, temp storage]
             cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &numsView,
             cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &a,
             cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &resView,
             cusp::array1d<FLOAT_TYPE, cusp::host_memory>::view &hView,
             int hIndex) {
  // cpu implementation
  cusp::blas::axpy(a, resView, -hView[hIndex]);
}

// subtract dots * prevVec from curVec using axpy
void do_axpy(hipblasHandle_t &cublasHandle,
             // numsView: [-1, 0, 1, temp storage]
             cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &numsView,
             cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &a,
             cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &resView,
             cusp::array1d<FLOAT_TYPE, cusp::device_memory>::view &hView,
             int hIndex) {
  // gpu implementation
  // cusp::blas::axpy(a, b, -hView[hIndex]);

  int count = a.size();

  double *minusOne = thrust::raw_pointer_cast(&numsView[0]);
  double *zero = thrust::raw_pointer_cast(&numsView[1]);
  double *temp = thrust::raw_pointer_cast(&numsView[3]);
  double *x = thrust::raw_pointer_cast(&a[0]);
  double *res = thrust::raw_pointer_cast(&resView[0]);
  double *h = thrust::raw_pointer_cast(&hView[hIndex]);

  // copy h to temp
  if (hipblasDcopy(cublasHandle, 1, h, 1, temp, 1) != HIPBLAS_STATUS_SUCCESS)
    error("hipblasDcopy() failed");

  // scale temp by -1
  if (hipblasDscal(cublasHandle, 1, minusOne, temp, 1) != HIPBLAS_STATUS_SUCCESS)
    error("hipblasDscal() failed");

  // do the axpy (alpha = temp)
  if (hipblasDaxpy(cublasHandle, count, temp, x, 1, res, 1) !=
      HIPBLAS_STATUS_SUCCESS)
    error("hipblasDaxpy() failed");
}

template <class MEMORY_TYPE> class CuspData {
  typedef cusp::array1d<FLOAT_TYPE, MEMORY_TYPE> Array1d;
  typedef typename Array1d::view Array1dView;

  typedef cusp::array2d<FLOAT_TYPE, MEMORY_TYPE> Array2d;
  typedef typename Array2d::view Array2dView;

  typedef cusp::array1d<FLOAT_TYPE, cusp::host_memory> HostFloatArray1d;
  typedef typename HostFloatArray1d::view HostFloatArray1dView;

  typedef cusp::array1d<int, cusp::host_memory> HostIntArray1d;
  typedef typename HostIntArray1d::view HostIntArray1dView;

  typedef cusp::coo_matrix<int, FLOAT_TYPE, MEMORY_TYPE> CooMatrix;

  typedef cusp::hyb_matrix<int, FLOAT_TYPE, MEMORY_TYPE> HybMatrix;

  typedef cusp::csr_matrix<int, FLOAT_TYPE, cusp::host_memory> HostCsrMatrix;
  typedef typename HostCsrMatrix::view HostCsrMatrixView;

private:
  HybMatrix *aMatrix;
  HybMatrix *keyDirMatrix;

  Array1d *vMatrix;    // p * [(i+1) * n]
  Array1d *hMatrix;    // p * [(i+1) * i]
  Array1d *vProjected; // p * [k * (i+1)]

  unsigned long _n; // number of dimensions in the system
  unsigned long _k; // number of key directions
  unsigned long _i; // number of arnoldi iterations
  unsigned long _p; // number of parallel initial vectors in arnoldi

  // profiling variables
  bool useProfiling;
  unsigned long aMatrixNonzeros;
  unsigned long keyDirMatrixNonzeros;

  // cublas variables
  hipblasHandle_t cublasHandle;
  Array1d *cuspNums; // [-1, 0, 1, temp_val]

public:
  GpuUtil util; // timers and other utility functions

  CuspData(bool useCpu) : util(useCpu), cublasHandle(0) {
    aMatrix = 0;
    keyDirMatrix = 0;

    vMatrix = 0;
    hMatrix = 0;
    vProjected = 0;

    cuspNums = 0;

    reset(); // this resets all variables
  }

  ~CuspData() {
    reset();
    hipDeviceReset();
  }

  void reset() {
    if (aMatrix != 0) {
      delete aMatrix;
      aMatrix = 0;
    }

    if (keyDirMatrix != 0) {
      delete keyDirMatrix;
      keyDirMatrix = 0;
    }

    if (vMatrix != 0) {
      delete vMatrix;
      vMatrix = 0;
    }

    if (hMatrix != 0) {
      delete hMatrix;
      hMatrix = 0;
    }

    if (vProjected != 0) {
      delete vProjected;
      vProjected = 0;
    }

    if (cuspNums != 0) {
      delete cuspNums;
      cuspNums = 0;
    }

    util.clearTimers();

    _n = 0;
    _k = 0;
    _i = 0;
    _p = 0;

    setUseProfiling(false);
    aMatrixNonzeros = 0;
    keyDirMatrixNonzeros = 0;
  }

  void setUseProfiling(bool enabled) {
    useProfiling = enabled;
    util.setUseProfiling(enabled);
  }

  // load A matrix, passed in as a csr matrix
  void loadAMatrix(unsigned long w, unsigned long h, int *rowOffsets,
                   unsigned long rowOffsetsLen, int *colInds,
                   unsigned long colIndsLen, FLOAT_TYPE *values,
                   unsigned long valuesLen) {
    if (_n == 0)
      error("loadKeyDirMatrix() called before preallocate() (_n==0)\n");

    if (w != h)
      error("loadAMatrix() expected square A matrix, got w=%lu, h=%lu", w, h);

    if (w != _n)
      error("in loadAMatrix() size (%lu) not to equal dims from preallocate() "
            "(%lu)",
            w, _n);

    if (useProfiling) {
      FLOAT_TYPE size = rowOffsetsLen * sizeof(int);
      size += colIndsLen * sizeof(int);
      size += valuesLen * sizeof(FLOAT_TYPE);

      printf("loadAMatrix() with sparse matrix size: %.2f MB (%lu nonzeros). "
             "Memory on device: "
             "%.2f MB\n",
             size / 1024.0 / 1024.0, valuesLen, getFreeMemoryMb());
    }

    aMatrixNonzeros = valuesLen;

    HostIntArray1dView rowOffsetsView(rowOffsets, rowOffsets + rowOffsetsLen);
    HostIntArray1dView colIndsView(colInds, colInds + colIndsLen);
    HostFloatArray1dView valuesView(values, values + colIndsLen);

    HostCsrMatrixView view(_n, _n, valuesLen, rowOffsetsView, colIndsView,
                           valuesView);

    if (aMatrix != 0) {
      delete aMatrix;
      aMatrix = 0;
    }

    try {
      aMatrix = new HybMatrix(view);
    } catch (std::bad_alloc) {
      error("memory allocation of aMatrix failed\n");
    }

    // one-time preallocate cusp_nums (shouldn't fail)
    if (cuspNums == 0) {
      try {
        HostFloatArray1d temp(4);
        temp[0] = -1;
        temp[1] = 0;
        temp[2] = 1;
        temp[3] = 0;
        cuspNums = new Array1d(temp);
      } catch (std::bad_alloc) {
        error("cuspNums memory allocation failed\n");
      }
    }
  }

  // load key dir matrix, passed in as a csr matrix
  void loadKeyDirMatrix(unsigned long w, unsigned long h, int *rowOffsets,
                        unsigned long rowOffsetsLen, int *colInds,
                        unsigned long colIndsLen, FLOAT_TYPE *values,
                        unsigned long valuesLen) {
    if (_n == 0)
      error("loadKeyDirMatrix() called before preallocate() (_n==0)\n");

    if (w != _n)
      error("in loadKeyDirMatrix() width (%lu) to equal dims from "
            "preallocate() (%lu)",
            w, _n);

    if (w != _n)
      error("in loadKeyDirMatrix() height (%lu) to equal keyDirMatSize from "
            "preallocate() "
            "(%lu)",
            h, _k);

    if (useProfiling) {
      FLOAT_TYPE size = rowOffsetsLen * sizeof(int);
      size += colIndsLen * sizeof(int);
      size += valuesLen * sizeof(FLOAT_TYPE);

      printf("loadKeyDirMatrix() with dense matrix size: %.2f MB, Memory on "
             "device: %.2f MB\n",
             size / 1024.0 / 1024.0, getFreeMemoryMb());
    }

    keyDirMatrixNonzeros = valuesLen;

    HostIntArray1dView rowOffsetsView(rowOffsets, rowOffsets + rowOffsetsLen);
    HostIntArray1dView colIndsView(colInds, colInds + colIndsLen);
    HostFloatArray1dView valuesView(values, values + colIndsLen);

    HostCsrMatrixView view(h, w, valuesLen, rowOffsetsView, colIndsView,
                           valuesView);

    if (keyDirMatrix != 0) {
      delete keyDirMatrix;
      keyDirMatrix = 0;
    }

    try {
      keyDirMatrix = new HybMatrix(view);
    } catch (std::bad_alloc) {
      error("memory allocation of keyDirMatrix() failed\n");
    }
  }

  FLOAT_TYPE getFreeMemoryMb() {
    unsigned long bytes = util.getFreeMemory();

    return bytes / 1024.0 / 1024.0;
  }

  // frees memory if it was previously allocated, returns false if memory error
  // occurs
  bool preallocateMemory(unsigned long arnoldiIt, unsigned long numParallelInit,
                         unsigned long dims, unsigned long keyDirMatSize) {
    bool success = true;

    if (_n != 0 && dims != _n && aMatrix != 0) {
      // num dimensions changed... free aMatrix
      delete aMatrix;
      aMatrix = 0;
    }

    if (_k != 0 && keyDirMatSize != _k && keyDirMatrix != 0) {
      // num key directions changed
      delete keyDirMatrix;
      keyDirMatrix = 0;
    }

    _n = dims;
    _i = arnoldiIt;
    _p = numParallelInit;
    _k = keyDirMatSize;

    if (useProfiling)
      printf("preallocateMemory() called with profiling, Free memory on "
             "device: %.2f MB\n",
             getFreeMemoryMb());

    // free memory which we will newly allocate

    if (hMatrix != 0) {
      delete hMatrix;
      hMatrix = 0;
    }

    if (vMatrix != 0) {
      delete vMatrix;
      vMatrix = 0;
    }

    if (vProjected != 0) {
      delete vProjected;
      vProjected = 0;
    }

    try {
      // preallocate hMatrix, numParInit * iterations * iterations
      unsigned long hMatrixSize = _p * _i * (_i + 1);

      if (useProfiling)
        printf("Trying to allocate %.2f MB for hMatrix (remaining memory %.2f "
               "MB)...\n",
               sizeof(FLOAT_TYPE) * hMatrixSize / 1024.0 / 1024.0,
               getFreeMemoryMb());

      hMatrix = new Array1d(hMatrixSize, 0);

      // preallocate vMatrix, width = dims * iterations, height = numParInit
      unsigned long vMatrixSize = _p * _n * (_i + 1);

      if (useProfiling)
        printf("Trying to allocate %.2f MB for vMatrix (remaining memory %.2f "
               "MB)...\n",
               sizeof(FLOAT_TYPE) * vMatrixSize / 1024.0 / 1024.0,
               getFreeMemoryMb());

      vMatrix = new Array1d(vMatrixSize, 0);

      // preallocate vProjected
      unsigned long vProjectedSize = _p * _k * (_i + 1);

      if (useProfiling)
        printf(
            "Trying to allocate %.2f MB for vProjected (remaining memory %.2f "
            "MB)...\n",
            sizeof(FLOAT_TYPE) * vProjectedSize / 1024.0 / 1024.0,
            getFreeMemoryMb());

      vProjected = new Array1d(vProjectedSize, 0);
    } catch (std::bad_alloc) {
      if (useProfiling)
        printf("memory allocation failed\n");

      _i = 0;
      _p = 0;
      _n = 0;
      _k = 0;
      success = false;
    }

    return success;
  }

  void initParallelArnoldi(unsigned long startDim, unsigned long numInitVecs) {
    util.tic("init parallel");

    if (startDim + numInitVecs > _n)
      error("initParallelArnoldiV called with startDim=%lu, numInitVecs=%lu, "
            "but dims=%lu",
            startDim, numInitVecs, _n);

    // fill with zeros
    cusp::blas::fill(*vMatrix, 0.0);

    unsigned long rowWidth = _n * (_i + 1);

    for (unsigned long rowNum = 0; rowNum < (unsigned long)numInitVecs;
         ++rowNum) {
      // initialize the "1.0" in each row
      unsigned long rowOffset = rowNum * rowWidth;

      (*vMatrix)[rowOffset + startDim + rowNum] = 1.0;
    }

    // also fill h with zeros
    cusp::blas::fill(*hMatrix, 0.0);

    // also fill projected vMatrix with zeros
    cusp::blas::fill(*vProjected, 0.0);

    util.toc("init parallel");
  }

  // reads/writes from/to vMatrix, writes to hMatrix
  void runArnoldi(unsigned long iterations, unsigned long numInitVecs) {
    Array1dView cuspNumsView = cuspNums->subarray(0, 4);

    // Arnoldi parallel algorithm iteration
    for (unsigned long it = 1; it <= iterations; it++) {
      // do all the multiplications up front
      util.tic("sparse matrix vector multiply");
      for (unsigned long curInitVec = 0; curInitVec < numInitVecs;
           ++curInitVec) {
        unsigned long pageOffset = curInitVec * _n * (iterations + 1);

        unsigned long prevRowOffset = _n * (it - 1);
        unsigned long curRowOffset = _n * it;

        Array1dView vecView = vMatrix->subarray(pageOffset + prevRowOffset, _n);
        Array1dView resultView =
            vMatrix->subarray(pageOffset + curRowOffset, _n);

        cusp::multiply(*aMatrix, vecView, resultView);
      }
      util.toc("sparse matrix vector multiply",
               2 * aMatrixNonzeros * numInitVecs);

      util.tic("dots & axpy");

      for (unsigned long curInitVec = 0; curInitVec < numInitVecs;
           ++curInitVec) {
        unsigned long pageOffset = curInitVec * (_i + 1) * _i;
        unsigned long rowOffset = (it - 1) * (_i + 1);
        Array1dView resultView = hMatrix->subarray(pageOffset + rowOffset, it);

        pageOffset = curInitVec * _n * (iterations + 1);
        rowOffset = _n * it;

        Array1dView curVec = vMatrix->subarray(pageOffset + rowOffset, _n);

        // combined dot/axpy to have modified gram-schmidt orthogonalization
        // (more stable)
        for (unsigned long row = 0; row < it; ++row) {
          Array1dView curRow = vMatrix->subarray(pageOffset + row * _n, _n);

          // util.tic("dot");
          dot_product(cublasHandle, _n, curVec, curRow, resultView, row);
          // util.toc("dot", 2 * _n);

          rowOffset = _n * row;
          Array1dView prevVec = vMatrix->subarray(pageOffset + rowOffset, _n);

          // util.tic("axpy");
          do_axpy(cublasHandle, cuspNumsView, prevVec, curVec, resultView, row);
          // util.toc("axpy", 2 * _n);
        }
      }

      util.toc("dots & axpy", 2 * 2 * _n * it * numInitVecs);

      util.tic("magnitude and scale");
      for (unsigned long curInitVec = 0; curInitVec < numInitVecs;
           ++curInitVec) {
        unsigned long pageOffset = curInitVec * _n * (iterations + 1);
        unsigned long rowOffset = _n * it;

        Array1dView curVec = vMatrix->subarray(pageOffset + rowOffset, _n);

        FLOAT_TYPE magnitude = cusp::blas::nrm2(curVec);

        // store magnitude in H
        pageOffset = curInitVec * (_i + 1) * _i;
        rowOffset = (it - 1) * (_i + 1);
        (*hMatrix)[pageOffset + rowOffset + it] = magnitude;

        // scale vector
        if (magnitude < 1e-10)
          cusp::blas::scal(curVec, 0.0);
        else
          cusp::blas::scal(curVec, 1.0 / magnitude);
      }
      util.toc("magnitude and scale");
    }
  }

  void projectV(unsigned long iterations, unsigned long numInitVecs) {
    // use vMatrix and keyDirMatrix to produce vProjected

    for (unsigned long iteration = 0; iteration <= iterations; ++iteration) {
      for (unsigned long curInitVec = 0; curInitVec < numInitVecs;
           ++curInitVec) {
        // Can we do this with a single matrix-matrix mult???

        unsigned long pageOffset = curInitVec * _n * (iterations + 1);
        unsigned long rowOffset = _n * (iteration);

        Array1dView vecView = vMatrix->subarray(pageOffset + rowOffset, _n);

        // result view is in vProjected
        pageOffset = curInitVec * _k * (iterations + 1);
        rowOffset = _k * iteration;

        Array1dView resultView =
            vProjected->subarray(pageOffset + rowOffset, iterations);

        util.tic("project-v sparse matrix vector multiply");
        cusp::multiply(*keyDirMatrix, vecView, resultView);
        util.toc("project-v sparse matrix vector multiply",
                 2 * keyDirMatrixNonzeros);
      }
    }
  }

  void printProfilingData() {
    util.printTimers();
    util.clearTimers();
  }

  void arnoldiParallel(unsigned long startDim, FLOAT_TYPE *resultH,
                       unsigned long sizeResultH, FLOAT_TYPE *resultPV,
                       unsigned long sizeResultPV) {
    if (aMatrix == 0)
      error("arnoldiParallel() called before loadAMatrix()\n");

    if (keyDirMatrix == 0)
      error("arnoldiParallel() called before loadKeyDirMatrix()\n");

    if (_i == 0 || _p == 0)
      error("arnoldiParallel() called before preallocate() (_i==0 or _p==0)\n");

    // check expected results sizes
    unsigned long expectedH = _p * _i * (_i + 1);
    unsigned long expectedPV = _p * (_i + 1) * _k;

    if (sizeResultH != expectedH)
      error("Wrong size for resultH with i = %lu. Got %lu, expected %lu.", _i,
            sizeResultH, expectedH);

    if (sizeResultPV != expectedPV)
      error("Wrong size for resultPV with (i, p, k) = (%lu, %lu, %lu). Got "
            "%lu, expected %lu.",
            _i, _p, _k, sizeResultPV, expectedPV);

    if (startDim >= _n)
      error("invalid startDim in arnoldi (%lu dim system): %lu", _n, startDim);

    util.tic("arnoldi parallel total");

    unsigned long parInitVecs = _p;

    if (startDim + parInitVecs > _n)
      parInitVecs = _n - startDim;

    initParallelArnoldi(startDim, parInitVecs);

    util.tic("runArnoldi()");
    runArnoldi(_i, parInitVecs);
    util.toc("runArnoldi()");

    // project v_matrix onto keyDirMatrix
    util.tic("projectV()");
    projectV(_i, parInitVecs);
    util.toc("projectV()");

    // copying H matrix to np.ndarray
    util.tic("copying H matrix to np.ndarray");
    HostFloatArray1dView hostHView(resultH, resultH + expectedH);
    cusp::blas::copy(*hMatrix, hostHView); // hostHView = *hMatrix
    util.toc("copying H matrix to np.ndarray");

    // copy vProjected to np.ndarray

    util.tic("copying V-projected matrix to np.ndarray");
    HostFloatArray1dView hostPVView(resultPV, resultPV + expectedPV);
    cusp::blas::copy(*vProjected, hostPVView); // hostPVView = *vProjected
    util.toc("copying V-projected matrix to np.ndarray");

    util.toc("arnoldi parallel total");
  }

private:
  void printV() {
    unsigned long h = _p * (_i + 1);
    unsigned long w = _n;

    cusp::print(
        make_array2d_view(h, w, w, Array1dView(*vMatrix), cusp::row_major()));
  }

  void printH() {
    unsigned long h = _p * (_i + 1);
    unsigned long w = _i;

    cusp::print(
        make_array2d_view(h, w, w, Array1dView(*hMatrix), cusp::row_major()));
  }
};

CuspData<cusp::host_memory> cuspDataCpu(true);
CuspData<cusp::device_memory> cuspDataGpu(false);

extern "C" {
unsigned long hasGpu() { return cuspDataGpu.util.hasGpu(); }

void reset() {
  cuspDataCpu.reset();
  cuspDataGpu.reset();
}

////// CPU Version
void setUseProfilingCpu(unsigned long enabled) {
  cuspDataCpu.setUseProfiling(enabled != 0);
}

// as csr matrix
void loadAMatrixCpu(unsigned long w, unsigned long h, int *rowOffsets,
                    unsigned long rowOffsetsLen, int *colInds,
                    unsigned long colIndsLen, FLOAT_TYPE *values,
                    unsigned long valuesLen) {
  cuspDataCpu.loadAMatrix(w, h, rowOffsets, rowOffsetsLen, colInds, colIndsLen,
                          values, valuesLen);
}

void loadKeyDirMatrixCpu(unsigned long w, unsigned long h, int *rowOffsets,
                         unsigned long rowOffsetsLen, int *colInds,
                         unsigned long colIndsLen, FLOAT_TYPE *values,
                         unsigned long valuesLen) {
  cuspDataCpu.loadKeyDirMatrix(w, h, rowOffsets, rowOffsetsLen, colInds,
                               colIndsLen, values, valuesLen);
}

FLOAT_TYPE getFreeMemoryMbCpu() { return cuspDataCpu.getFreeMemoryMb(); }

unsigned long preallocateMemoryCpu(unsigned long arnoldiIt,
                                   unsigned long numParallelInitVecs,
                                   unsigned long dims,
                                   unsigned long keyDirMatSize) {
  return cuspDataCpu.preallocateMemory(arnoldiIt, numParallelInitVecs, dims,
                                       keyDirMatSize)
             ? 1
             : 0;
}

void arnoldiParallelCpu(unsigned long startDim, FLOAT_TYPE *resultH,
                        unsigned long sizeResultH, FLOAT_TYPE *resultPV,
                        unsigned long sizeResultPV) {
  cuspDataCpu.arnoldiParallel(startDim, resultH, sizeResultH, resultPV,
                              sizeResultPV);
}

void printProfilingDataCpu() { cuspDataCpu.printProfilingData(); }

////// GPU Version
void setUseProfilingGpu(unsigned long enabled) {
  cuspDataGpu.setUseProfiling(enabled != 0);
}

// as csr matrix
void loadAMatrixGpu(unsigned long w, unsigned long h, int *rowOffsets,
                    unsigned long rowOffsetsLen, int *colInds,
                    unsigned long colIndsLen, FLOAT_TYPE *values,
                    unsigned long valuesLen) {
  cuspDataGpu.loadAMatrix(w, h, rowOffsets, rowOffsetsLen, colInds, colIndsLen,
                          values, valuesLen);
}

// as csr matrix
void loadKeyDirMatrixGpu(unsigned long w, unsigned long h, int *rowOffsets,
                         unsigned long rowOffsetsLen, int *colInds,
                         unsigned long colIndsLen, FLOAT_TYPE *values,
                         unsigned long valuesLen) {
  cuspDataGpu.loadKeyDirMatrix(w, h, rowOffsets, rowOffsetsLen, colInds,
                               colIndsLen, values, valuesLen);
}

FLOAT_TYPE getFreeMemoryMbGpu() { return cuspDataGpu.getFreeMemoryMb(); }

unsigned long preallocateMemoryGpu(unsigned long arnoldiIterations,
                                   unsigned long numParallelInitVecs,
                                   unsigned long dims,
                                   unsigned long keyDirMatSize) {
  return cuspDataGpu.preallocateMemory(arnoldiIterations, numParallelInitVecs,
                                       dims, keyDirMatSize)
             ? 1
             : 0;
}

void arnoldiParallelGpu(unsigned long startDim, FLOAT_TYPE *resultH,
                        unsigned long sizeResultH, FLOAT_TYPE *resultPV,
                        unsigned long sizeResultPV) {
  cuspDataGpu.arnoldiParallel(startDim, resultH, sizeResultH, resultPV,
                              sizeResultPV);
}

void printProfilingDataGpu() { cuspDataGpu.printProfilingData(); }

} // end extern "C"
