#include "hip/hip_runtime.h"
// Dung Tran
// An interface
// Krylov subspace - based simulation using Gpu- Cusp / Cuda for sparse ode
// June 2017

#include <new>
#include <stdio.h>
#include <stdlib.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/multiply.h>
#include <cusp/print.h>
#include <cusp/blas.h>
#include <sys/time.h>

typedef double FLOAT_TYPE;
typedef cusp::host_memory MEMORY_TYPE;
//typedef cusp::device_memory MEMORY_TYPE;

//static int choose_GPU = 0; // choose_GPU == 1 means that user choose to use GPU, if not, using CPU
// shared matrix in device memory
static cusp::hyb_matrix<int, FLOAT_TYPE, MEMORY_TYPE>* curMatrix = 0;
static std::vector< cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> > V_;
static std::vector< cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> > V_all; // use to compute n- Vm matrix
static std::vector< cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> > V_all_final; // contain all n- Vm matrix
static std::vector< cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> > H_all; // contain all n Hm matrix
static std::vector< cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> > expHt_all; // contain all n exp(H*t) matrix, used to compute simulation result


static std::vector< cusp::array1d<FLOAT_TYPE, MEMORY_TYPE> > device_sim_result;
static cusp::hyb_matrix<int, FLOAT_TYPE, MEMORY_TYPE>* keyDirMatrix = 0;
static int numStepOfSim = 0;
static int systemSize = 0;
static int keyDirMatrix_w = 0;
static int keyDirMatrix_h = 0;

// timing shared variable
static long lastTicUs = 0;


void _choose_GPU_or_CPU(char* msg)
{
    if (strcmp(msg, "CPU")){
            printf("User choosed to use CPU \n");
            // choose_GPU = 0;
        }
        else if(strcmp(msg,"GPU")){
            printf("User choosed to use GPU \n");
            // choose_GPU = 1;
        }  else{
            printf("wrong input options, choose CPU or GPU only \n");
        }

}
 


void error(const char* msg)
{
    printf("Fatal Error: %s\n", msg);
    exit(1);             
}

void tic()
{
    struct timeval now;
    
    if(gettimeofday( &now, 0))
        error("gettimeofday");
        
    lastTicUs = 1000000 * now.tv_sec + now.tv_usec;
}

// returns the us elaspsed
long toc(const char* label)
{
    struct timeval now;
    
    if(gettimeofday( &now, 0))
        error("gettimeofday");
        
    long nowUs = 1000000 * now.tv_sec + now.tv_usec;
    long dif = nowUs - lastTicUs;
    
    printf("%s: %.4f ms\n", label, dif / 1000.0);
    
    return dif;
}

void _loadMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{
    tic();
    cusp::coo_matrix<int, FLOAT_TYPE, cusp::host_memory> hostMatrix(w, h, nonZeroCount);
        
    printf("loadMatrix() called, estimated size in memory of sparse matrix: %.2f MB (%d nonzeros)\n", 
        nonZeroCount * (8 + 4 + 4) / 1024.0 / 1024.0, nonZeroCount);

    // initialize matrix entries on host
    int index = 0;
 
    for (int i = 0; i < nonZeroCount; ++i)
    {
        int row = nonZeroRows[i];
        int col = nonZeroCols[i];
        double val = nonZeroEntries[i];
        
        hostMatrix.row_indices[index] = row;
        hostMatrix.column_indices[index] = col;
        hostMatrix.values[index++] = val;
    }
    
    toc("creating host coo matrix");
    
    tic();
    if (curMatrix != 0)
    {
        delete curMatrix;
        curMatrix = 0;
    }
    
    curMatrix = new (std::nothrow) cusp::hyb_matrix<int, FLOAT_TYPE,MEMORY_TYPE>(hostMatrix);
        
    if (curMatrix == 0)
        error("allocation of heap-based csr matrix in device memory returned nullptr");
        
    toc("copying matrix to device memory");
}

void _loadKeyDirMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{   // Load key Direction Sparse Matrix to get a particular direction of simulation result
    
    tic();
    keyDirMatrix_w = w;
    keyDirMatrix_h = h;
    
    cusp::coo_matrix<int, FLOAT_TYPE, cusp::host_memory> hostKeyMatrix(w, h, nonZeroCount);
        
    printf("loadKeyDirMatrix() called, estimated size in memory of sparse matrix: %.2f MB (%d nonzeros)\n", 
        nonZeroCount * (8 + 4 + 4) / 1024.0 / 1024.0, nonZeroCount);

    // initialize key matrix entries on host
    int index = 0;
 
    for (int i = 0; i < nonZeroCount; ++i)
    {
        int row = nonZeroRows[i];
        int col = nonZeroCols[i];
        double val = nonZeroEntries[i];
        
        hostKeyMatrix.row_indices[index] = row;
        hostKeyMatrix.column_indices[index] = col;
        hostKeyMatrix.values[index++] = val;
    }
    
    toc("creating host coo key matrix");
    
    tic();
    if (keyDirMatrix != 0)
    {
        delete keyDirMatrix;
        keyDirMatrix = 0;
    }
    
    keyDirMatrix = new (std::nothrow) cusp::hyb_matrix<int, FLOAT_TYPE,MEMORY_TYPE>(hostKeyMatrix);
        
    if (keyDirMatrix == 0)
        error("allocation of heap-based csr key matrix in device memory returned nullptr");
        
    toc("copying key matrix to device memory");

}

int _arnoldi_initVector(double* init_vector, double* result_H, int size, int numIter)
{
    
    if (curMatrix == 0)
        error("loadMatrix must be called before running arnoldi algorithm");
    
    // initialize input vector
    tic();
    cusp::array1d<FLOAT_TYPE, cusp::host_memory> hostInitVec(size);
    
    for (int i = 0; i < size; ++i)
        hostInitVec[i] = init_vector[i];
    toc("creating hostVec initial vector");
    
    // copy initial vector to device memory
    tic();
    cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> deviceInitVec(hostInitVec);
    toc("copying initial vetocctor to device memory");

    // system dimension 
    tic();
    int N = size;
    toc("get system dimension");

    // maximum number of Iteration of Arnoldi algorithm
    tic();
    int maxiter = std::min(N, numIter);
    toc("get maximum number of iteration of arnoldi algorithm");

    // create matrix H_ in device memory for iteration
    tic();	
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H_(maxiter + 1, maxiter, 0);
    toc("create matrix H_ in device memory for iteration");

    // returned matrix H after iteration -- Hm in the algorithm -- (m x m) matrix 
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H(maxiter, maxiter); 
    toc("create returned matrix H after iteration -- Hm in the algorithm -- (m x m) matrix ");

    // create matrix V_ for iteration
    tic();
    V_.resize(maxiter+1);
    for (int i = 0; i < maxiter + 1; i++)
        V_[i].resize(N);
    toc("create matrix V_ for iteration");

    // copy initial vector into V_[0]
    tic(); 
    cusp::copy(deviceInitVec,V_[0]); 
    toc("copy initial vector into V_[0]"); 

    // compute beta 
    tic();
    FLOAT_TYPE beta = cusp::blas::nrm2(deviceInitVec);
    toc("compute beta");
   

    // normalize initial vector
    cusp::blas::scal(V_[0], float(1)/beta);

    // iteration
    tic();
    int j;
    for(j = 0; j < maxiter; j++)
    {
	cusp::multiply(*curMatrix, V_[j], V_[j + 1]);
	
	for(int i = 0; i <= j; i++)
	{
		H_(i,j) = cusp::blas::dot(V_[i], V_[j + 1]);

		cusp::blas::axpy(V_[i], V_[j + 1], -H_(i,j));
	}

		H_(j+1,j) = cusp::blas::nrm2(V_[j + 1]);

		if(H_(j+1,j) < 1e-10) break;

		cusp::blas::scal(V_[j + 1], float(1) / H_(j+1,j));

     }
     toc("iteration");


     // scale V_ with beta, i.e. beta*V_, used later for computing simulation trace
     tic();
     for(int i = 0; i < maxiter; i++)
     {
        cusp::blas::scal(V_[i],beta);
     }
     toc("scaling matrix V with beta");
     

     // get matrix H (m x m dimension)
     tic(); 
     for(int rowH=0;rowH < maxiter; rowH++)
     for(int colH = 0; colH <maxiter; colH++)
		H(rowH,colH) = H_(rowH,colH);
     toc("get matrix H -- (m x m) dimension");


     // copying H matrix to np.ndarray
     tic();
    
     for (int i = 0; i < numIter; ++i )
	    for (int k = 0; k < numIter; ++k)
		    result_H[i*numIter + k] = H_(i,k);       
     toc("copying H to np.ndarray");
     
     if(j < maxiter)
     return j+1;
     else return maxiter;
}

int _arnoldi_initVectorPos(int basic_initVector_pos, double* result_H, int size, int numIter)
{   
    if (curMatrix == 0)
        error("loadMatrix must be called before running arnoldi algorithm");
    
    // create initial basic vector on device memory
    tic();

    cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> deviceInitVec(size,0);

    deviceInitVec[basic_initVector_pos] = 1;
    toc("create initial basic vector on device memory based on its position, i.e. basic_initVector_pos");
    
    // system dimension 
    tic();
    int N = size;
    toc("get system dimension");

    // maximum number of Iteration of Arnoldi algorithm
    tic();
    int maxiter = std::min(N, numIter);
    toc("get maximum number of iteration of arnoldi algorithm");

    // create matrix H_ in device memory for iteration
    tic();	
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H_(maxiter + 1, maxiter, 0);
    toc("create matrix H_ in device memory for iteration");

    // returned matrix H after iteration -- Hm in the algorithm -- (m x m) matrix 
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H(maxiter, maxiter); 
    toc("create returned matrix H after iteration -- Hm in the algorithm -- (m x m) matrix ");

    // create matrix V_ for iteration
    tic();
    V_.resize(maxiter+1);
    for (int i = 0; i < maxiter + 1; i++)
        V_[i].resize(N);
    toc("create matrix V_ for iteration");

    // copy initial vector into V_[0]
    tic(); 
    cusp::copy(deviceInitVec,V_[0]); 
    toc("copy initial vector into V_[0]"); 

    // iteration
    tic();
    int j;
    for(j = 0; j < maxiter; j++)
    {
	cusp::multiply(*curMatrix, V_[j], V_[j + 1]);
    
	for(int i = 0; i <= j; i++)
	{
		H_(i,j) = cusp::blas::dot(V_[i], V_[j + 1]);

		cusp::blas::axpy(V_[i], V_[j + 1], -H_(i,j));
	}

		H_(j+1,j) = cusp::blas::nrm2(V_[j + 1]);

		if(H_(j+1,j) < 1e-10) break;

		cusp::blas::scal(V_[j + 1], float(1) / H_(j+1,j));

     }
     toc("iteration time of Arnoldi algorithm");
         
     // get matrix H (m x m dimension)
     tic(); 
     for(int rowH=0;rowH < maxiter; rowH++)
	    for(int colH = 0; colH <maxiter; colH++)
		    H(rowH,colH) = H_(rowH,colH);
     toc("get matrix H -- (m x m) dimension");


     // copying H matrix to np.ndarray
     tic();
    
     for (int i = 0; i < numIter; ++i )
	    for (int k = 0; k < numIter; ++k)
		    result_H[i*numIter + k] = H_(i,k);       
     toc("copying H to np.ndarray");

     if(j < maxiter)
     return j+1;
     else return maxiter;
}


int _arnoldi_parallel(int size, int numIter,double* result_H)
{   
    if (curMatrix == 0)
        error("loadMatrix must be called before running arnoldi algorithm");
    
    // maximum number of Iteration of Arnoldi algorithm
    tic();
    int maxiter = std::min(size, numIter);
    toc("get maximum number of iteration of arnoldi algorithm");

    // create matrix V_all to contain all matrix V: V_all = [V0 V1 ...Vm]
    // V0 = [V0_1 ... V0_n] is (n x n) matrix containing all initial vectors of n-dimensions system
    // Vi = [Vi_1 ... Vi_n] is (n x n) matrix containing all i-th vectors in step i of Arnoldi algorithm
    
    tic();
    V_all.resize(maxiter+1);
    toc("create matrix V_all to contain all matrix Vm");

    // create matrix V_all_final to contain all matrix V; V_all_final = [Vm_0 Vm_2 ...Vm_(n-1)]
    // Vm_0 is the matrix (n x m) V (obtained from Arnoldi algorithm) that corresponds to the 0-th initial vector  
    // Vm_i is the (n x m) matrix V (obtained from Arnoldi algorithm) that corresponds to the i-th initial vector
    
    tic();
    V_all_final.resize(size);
    toc("create matrix V_all_final to contain all matrix Vm");    

    // create matrix H_all to contain all matrix H: H_all = [Hm_1 Hm_2 ...Hm_n]
    // Hm_1, Hm_2 , ... Hm_n are m x m matrices, Hm_i is conresponding to the initial vector i 
        
    tic();
    H_all.resize(size+1);
    toc("create matrix H_all to contain all matrix H");

     // create initial basic vector V_all[0] = n-dimension identity mat
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> identity_mat(size,size,0);
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> zero_Vmat(size,size,0);
    cusp::array2d<FLOAT_TYPE, MEMORY_TYPE> Hmat_k(maxiter+1,maxiter,0);
    
   
    for (int i = 0; i < size; i++)
        for(int j = 0; j < size; j++)
            if (i == j) identity_mat(i,j) = 1;
    
    for (int i = 0; i< size; i++)
        cusp::copy(Hmat_k,H_all[i]); // initialize H_all[i] 

    for (int i = 1; i < maxiter+1; i++)
        cusp::copy(zero_Vmat,V_all[i]); // initalize V_all[i]

    cusp::copy(identity_mat,V_all[0]); // initialize V_all[0] by basic initial vectors
    
    toc("initialize V_all and H_all, create all initial basic vector on device memory V_all[0]");
    
    // Arnoldi parallel algorithm iteration

 
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> Vj_plus1(size,size,0);
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> Vi(size,size,0);
    cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> Vj_plus1_col_k(size); 
    cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> Vi_col_k(size);
    
    tic();
    int j;
    for (j = 0; j < maxiter; j++){

        cusp::multiply(*curMatrix,V_all[j],V_all[j+1]);
         
        cusp::copy(V_all[j+1],Vj_plus1);
  
        for(int k = 0; k < size; k++){
            // compute Hm-k 

            for(int l = 0; l < size; l++){
                Vj_plus1_col_k[l] = Vj_plus1(l,k); // Load column k of Vj_plus1
            }

            cusp::copy(H_all[k],Hmat_k); // Load k-th Hmat matrix         
                
            for(int i = 0; i <= j; i++){
                
                cusp::copy(V_all[i],Vi);

                for(int l = 0; l < size; l++){

                    Vi_col_k[l] = Vi(l,k); // Load column k of Vi
                    
                }
                

                Hmat_k(i,j) = cusp::blas::dot(Vi_col_k, Vj_plus1_col_k);
                
                cusp::blas::axpy(Vi_col_k, Vj_plus1_col_k, -Hmat_k(i,j));

            }
            
            Hmat_k(j+1,j) = cusp::blas::nrm2(Vj_plus1_col_k);

		    if(Hmat_k(j+1,j) < 1e-10) break;

		    cusp::blas::scal(Vj_plus1_col_k, float(1) / Hmat_k(j+1,j));
            
            for(int l = 0; l < size; l++){
                Vj_plus1(l,k) =  Vj_plus1_col_k[l]; // update  column k of Vj_plus1
            }

            cusp::copy(Hmat_k,H_all[k]); // update the k-th Hmatrix
            
        }

        cusp::copy(Vj_plus1,V_all[j+1]); // update column k of  V_all[j+1], i.e. corresponding to the k-th initial vector

    }
    
    toc("iteration time of Arnoldi algorithm");

     // copying H matrix to np.ndarray
     tic();
    
      // copying H matrix to np.ndarray
     tic();
     cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H(maxiter+1,maxiter,0);
     for (int k = 0; k< size; ++k){   
         cusp::copy(H_all[k],H);
         for (int i = 0; i < numIter; ++i){
             for(int l = 0; l < numIter; ++l)
                 result_H[i*numIter + l + k*numIter*numIter] = H(i,l);
         }
     }
          
     toc("copying H matrix to np.ndarray");

     int actual_numIter = 0; 

    // return actual number of iteration     
    if(j < maxiter)
    actual_numIter = j+1;
    else actual_numIter = maxiter;

    
     // save all matrix Vm into V_all_final
     tic();
     cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> Vm(size,actual_numIter,0); 
    
     
     for (int k = 0; k < size; k++){

         for(int i = 0; i < actual_numIter; i++){

             cusp::copy(V_all[i],Vi);

             for (int l = 0; l < size; l++){

                 Vm(l,i) = Vi(l,k); // fill the column i- of Vm by the column k of V                 
             }    
         }
         cusp::copy(Vm, V_all_final[k]);

     }   
     
     toc("save all matrix Vm into V_all_final");
     

    return actual_numIter;
    
}


void _sim(double* matrix_Hf, double* sim_result, int size, int actual_numIter, int numStep)
{
    // compute the simulation result and copy the result back to the CPU (in the sim_result variable)  	  
    // copy matrix Hf to device memory
    numStepOfSim = numStep;
    systemSize   = size;
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> deviceMatrix_Hf(actual_numIter,numStep);
    for(int k=0; k< numStep ; k++)
       for(int i = 0; i < actual_numIter; i++)
          deviceMatrix_Hf(i,k) = matrix_Hf[i*numStep+k];
    toc("copying matrix Hf to device memory");

    // compute simulation result 
    tic();
    device_sim_result.resize(numStep);
    for (int i = 0; i < numStep; i++)
        device_sim_result[i].resize(size);        
    toc("create simulation result matrix");

    tic();
    for (int i=0; i< numStep; i++)
    {
        for(int k=0; k < actual_numIter; k++)
        {
	      cusp::blas::axpy(V_[k], device_sim_result[i], deviceMatrix_Hf(k,i));
        }

    }
    toc("compute simulation result");

    // copy simulation result to np.ndarray
    tic();
    for(int i = 0; i < numStep; i++)
       for(int k = 0; k < size; k++)
       {
		sim_result[i*numStep + k] = device_sim_result[i][k];		
		
       }
    toc("copy simulation result to np.ndarray");
           
}

void _sim2(double* matrix_Hf, int size, int actual_numIter, int numStep)
{
    numStepOfSim = numStep;
    systemSize   = size;
    // compute simulation result and save on the device memory. Do not return the simulation result back to CPU     	  
    // copy matrix Hf to device memory
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> deviceMatrix_Hf(actual_numIter,numStep);
    for(int k=0; k< numStep ; k++)
       for(int i = 0; i < actual_numIter; i++)
          deviceMatrix_Hf(i,k) = matrix_Hf[i*numStep+k];
    toc("copying matrix Hf to device memory");

    // compute simulation result 
    tic();
    device_sim_result.resize(numStep);
    for (int i = 0; i < numStep; i++)
        device_sim_result[i].resize(size);        
    toc("create simulation result matrix");

    tic();
    for (int i=0; i< numStep; i++)
    {
        for(int k=0; k < actual_numIter; k++)
        {
	      cusp::blas::axpy(V_[k], device_sim_result[i], deviceMatrix_Hf(k,i));
        }

    }
    toc("compute simulation result without copy the result back to CPU"); 
}


void _getKeySimResult(double* keySimResult)
{   // Get the simulation result in a particular dimension, where the dimension is specified by a sparse matrix
    // Steps for using this function:
    // Step1. Load the direction sparse matrix by calling _loadKeyDirMatrix() function
    // Step2. Run the arnoldi algorithm by calling arnoldi_initVector() or arnoldi_initVectorPos()
    // Step3. Compute the matrix Hf = exp(i*timeStep*Hm)
    // Step4. Call _sim2() to compute the simulation result. This function saves the result on device memory and doesnot copy the result to the CPU
    // Step5. Call this function to get the simulation result at this particular dimension

    tic();
    // check consistency and compute key simulation result
    if (numStepOfSim == 0 || systemSize == 0) // check if there is simulation result in device memory
        printf("\n There is no simulation result. Call _sim2() method first");
    else
    {   printf("\n Number of simulation step = %d",numStepOfSim);
        printf("\n Number of Rows of key direction matrix = %d", keyDirMatrix_w);
        if (keyDirMatrix_h != systemSize) // check consistency between the key direction matrix and system dimension
        {
             printf("\n The number of column of key direction matrix is inconsistent with the system dimension");
             toc("check consistency");
        }   
        else
        {   // create key simulation result in device memory
            tic();
            std::vector< cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> > device_keySimResult(numStepOfSim);
            for (int i = 0; i < numStepOfSim; i++)
                device_keySimResult[i].resize(keyDirMatrix_w);
            toc("\n Create keySimResult in device memory");
            
            // compute key simulation result
            tic();
            
            for(int i = 0; i <numStepOfSim; i++)
            {
                cusp::multiply(*keyDirMatrix, device_sim_result[i],device_keySimResult[i]);
            }
            toc("\n Compute key simulation result");

            // copy key direction simulation result to np.array
            tic();
            for(int i = 0; i < numStepOfSim; i++)
                for(int k = 0; k < keyDirMatrix_w; k++)
                   {
		                keySimResult[i*numStepOfSim + k] = device_keySimResult[i][k];		
                   }
            toc("\n Copy key simulation result to np.array");
               
        }
        
    } 
    
}

void _getKeySimResult_parallel(int size, int H_numRows, double* expHt_tuples)
{
    // get Simulation result in specific direction defined by keyDirMatrix
    // SimResult = V*exp(H*t)*e1, (V,H) are matrices obtained from Arnoldi algorithm
    // KeySimResult = keyDirMatrix*SimResult

    // Check consitency 
    if (keyDirMatrix_h != size) // check consistency between the key direction matrix and system dimension
        {
             printf("\n The number of column of key direction matrix is inconsistent with the system dimension");
             toc("check consistency");
        }
    else{
        tic();
        expHt_all.resize(size);
        cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> device_expHt(H_numRows,H_numRows,0);

        for (int i = 0; i < size; i++){           
            for(int k=0; k< H_numRows ; k++)
               for(int l = 0; l < H_numRows; l++)
                  device_expHt(k,l) = expHt_tuples[k*H_numRows+l+i*H_numRows*H_numRows];

            cusp::copy(device_expHt,expHt_all[i]);

            printf("The %d-th exp(Ht) matrix is: \n",i);
            cusp::print(expHt_all[i]);
            
        }

        toc("copying exp(H*t) into device memory"); 

      }
    
    
}

int _hasGpu()
{
    int rv = 1;
    
    try
    {
        cusp::array1d<FLOAT_TYPE, cusp::host_memory> hostVec(10);
    
        for (int i = 0; i < 10; ++i)
            hostVec[i] = 0;

        cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> deviceVec(hostVec);
    }
    catch(std::exception &e)
    {
        printf("hasGpu() Failed: %s\n", e.what());
        rv = 0;
    }
    
    return rv;
}

extern "C"
{
int hasGpu()
{
    return _hasGpu();
}

void choose_GPU_or_CPU(char* msg)
{
    _choose_GPU_or_CPU(msg);
}

void loadMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{
    _loadMatrix(w, h, nonZeroRows, nonZeroCols, nonZeroEntries, nonZeroCount);
}


void loadKeyDirMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{
    _loadKeyDirMatrix(w, h, nonZeroRows, nonZeroCols, nonZeroEntries, nonZeroCount);
}
    
int arnoldi_initVector(double* init_vector, double* result_H, int size, int numIter)
{
   return _arnoldi_initVector(init_vector, result_H, size, numIter);
   
}

int arnoldi_initVectorPos(int basic_initVector_pos, double* result_H, int size, int numIter)
{
   return _arnoldi_initVectorPos(basic_initVector_pos, result_H, size, numIter);
}

    int arnoldi_parallel(int size, int numIter, double* result_H)
{
    return _arnoldi_parallel(size, numIter,result_H);
}

void sim(double* matrix_Hf, double* sim_result, int size, int actual_numIter, int numStep)\
{
    _sim(matrix_Hf, sim_result,  size, actual_numIter, numStep);
}
    
void sim2(double* matrix_Hf, int size, int actual_numIter, int numStep)\
{
    _sim2(matrix_Hf, size, actual_numIter, numStep);

}

void getKeySimResult(double* keySimResult)
{
    _getKeySimResult(keySimResult);
}

    
void getKeySimResult_parallel(int size, int numIter, double* expHt_tuples)
{
    _getKeySimResult_parallel(size,numIter,expHt_tuples);   
}

}
