#include "hip/hip_runtime.h"
// Dung Tran
// An interface
// Krylov subspace - based simulation using Gpu- Cusp / Cuda for sparse ode
// June 2017

#include <new>
#include <stdio.h>
#include <stdlib.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/multiply.h>
#include <cusp/print.h>
#include <cusp/blas.h>
#include <sys/time.h>

typedef double FLOAT_TYPE;
typedef cusp::host_memory MEMORY_TYPE; // using CPU for computation

//static int choose_GPU = 0; // choose_GPU == 1 means that user choose to use GPU, if not, using CPU
// shared matrix in device memory
static cusp::hyb_matrix<int, FLOAT_TYPE, MEMORY_TYPE>* curMatrix = 0;
static std::vector< cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> > V_;
static std::vector< cusp::array2d<FLOAT_TYPE,MEMORY_TYPE, cusp::column_major> > V_all; // use to compute n- Vm matrix in parallel
static std::vector< cusp::array2d<FLOAT_TYPE,MEMORY_TYPE, cusp::column_major> > V_all_final; // contain all n- Vm matrix
static std::vector< cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> > H_all; // contain all n Hm matrix


static std::vector< cusp::array1d<FLOAT_TYPE, MEMORY_TYPE> > device_sim_result;
static std::vector< cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> >  device_keySimResult_tuples; // contain all keySimResult
static cusp::hyb_matrix<int, FLOAT_TYPE, MEMORY_TYPE>* keyDirMatrix = 0;
static int numStepOfSim = 0;
static int systemSize = 0;
static int keyDirMatrix_w = 0;
static int keyDirMatrix_h = 0;

// timing shared variable
static long lastTicUs = 0;


void _choose_GPU_or_CPU(char* msg)
{
    printf("User choose to use %s for computation \n",msg);
    
}
 


void error(const char* msg)
{
    printf("Fatal Error: %s\n", msg);
    exit(1);             
}

void tic()
{
    struct timeval now;
    
    if(gettimeofday( &now, 0))
        error("gettimeofday");
        
    lastTicUs = 1000000 * now.tv_sec + now.tv_usec;
}

// returns the us elaspsed
long toc(const char* label)
{
    struct timeval now;
    
    if(gettimeofday( &now, 0))
        error("gettimeofday");
        
    long nowUs = 1000000 * now.tv_sec + now.tv_usec;
    long dif = nowUs - lastTicUs;
    
    printf("%s: %.4f ms\n", label, dif / 1000.0);
    
    return dif;
}

void _loadMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{
    tic();
    cusp::coo_matrix<int, FLOAT_TYPE, cusp::host_memory> hostMatrix(w, h, nonZeroCount);
        
    printf("loadMatrix() called, estimated size in memory of sparse matrix: %.2f MB (%d nonzeros)\n", 
        nonZeroCount * (8 + 4 + 4) / 1024.0 / 1024.0, nonZeroCount);

    // initialize matrix entries on host
    int index = 0;
 
    for (int i = 0; i < nonZeroCount; ++i)
    {
        int row = nonZeroRows[i];
        int col = nonZeroCols[i];
        double val = nonZeroEntries[i];
        
        hostMatrix.row_indices[index] = row;
        hostMatrix.column_indices[index] = col;
        hostMatrix.values[index++] = val;
    }
    
    toc("creating host coo matrix");
    
    tic();
    if (curMatrix != 0)
    {
        delete curMatrix;
        curMatrix = 0;
    }
    
    curMatrix = new (std::nothrow) cusp::hyb_matrix<int, FLOAT_TYPE,MEMORY_TYPE>(hostMatrix);
        
    if (curMatrix == 0)
        error("allocation of heap-based csr matrix in device memory returned nullptr");
        
    toc("copying matrix to device memory");
}

void _loadKeyDirMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{   // Load key Direction Sparse Matrix to get a particular direction of simulation result
    
    tic();
    keyDirMatrix_w = w;
    keyDirMatrix_h = h;
    
    cusp::coo_matrix<int, FLOAT_TYPE, cusp::host_memory> hostKeyMatrix(w, h, nonZeroCount);
        
    printf("loadKeyDirMatrix() called, estimated size in memory of sparse matrix: %.2f MB (%d nonzeros)\n", 
        nonZeroCount * (8 + 4 + 4) / 1024.0 / 1024.0, nonZeroCount);

    // initialize key matrix entries on host
    int index = 0;
 
    for (int i = 0; i < nonZeroCount; ++i)
    {
        int row = nonZeroRows[i];
        int col = nonZeroCols[i];
        double val = nonZeroEntries[i];
        
        hostKeyMatrix.row_indices[index] = row;
        hostKeyMatrix.column_indices[index] = col;
        hostKeyMatrix.values[index++] = val;
    }
    
    toc("creating host coo key matrix");
    
    tic();
    if (keyDirMatrix != 0)
    {
        delete keyDirMatrix;
        keyDirMatrix = 0;
    }
    
    keyDirMatrix = new (std::nothrow) cusp::hyb_matrix<int, FLOAT_TYPE,MEMORY_TYPE>(hostKeyMatrix);
        
    if (keyDirMatrix == 0)
        error("allocation of heap-based csr key matrix in device memory returned nullptr");
        
    toc("copying key matrix to device memory");

}

int _arnoldi_initVector(double* init_vector, double* result_H, int size, int numIter)
{
    
    if (curMatrix == 0)
        error("loadMatrix must be called before running arnoldi algorithm");
    
    // initialize input vector
    tic();
    cusp::array1d<FLOAT_TYPE, cusp::host_memory> hostInitVec(size);
    
    for (int i = 0; i < size; ++i)
        hostInitVec[i] = init_vector[i];
    toc("creating hostVec initial vector");
    
    // copy initial vector to device memory
    tic();
    cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> deviceInitVec(hostInitVec);
    toc("copying initial vetocctor to device memory");

    // system dimension 
    tic();
    int N = size;
    toc("get system dimension");

    // maximum number of Iteration of Arnoldi algorithm
    tic();
    int maxiter = std::min(N, numIter);
    toc("get maximum number of iteration of arnoldi algorithm");

    // create matrix H_ in device memory for iteration
    tic();	
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H_(maxiter + 1, maxiter, 0);
    toc("create matrix H_ in device memory for iteration");

    // returned matrix H after iteration -- Hm in the algorithm -- (m x m) matrix 
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H(maxiter, maxiter); 
    toc("create returned matrix H after iteration -- Hm in the algorithm -- (m x m) matrix ");

    // create matrix V_ for iteration
    tic();
    V_.resize(maxiter+1);
    for (int i = 0; i < maxiter + 1; i++)
        V_[i].resize(N);
    toc("create matrix V_ for iteration");

    // copy initial vector into V_[0]
    tic(); 
    cusp::copy(deviceInitVec,V_[0]); 
    toc("copy initial vector into V_[0]"); 

    // compute beta 
    tic();
    FLOAT_TYPE beta = cusp::blas::nrm2(deviceInitVec);
    toc("compute beta");
   

    // normalize initial vector
    cusp::blas::scal(V_[0], float(1)/beta);

    // iteration
    tic();
    int j;
    for(j = 0; j < maxiter; j++)
    {
	cusp::multiply(*curMatrix, V_[j], V_[j + 1]);
	
	for(int i = 0; i <= j; i++)
	{
		H_(i,j) = cusp::blas::dot(V_[i], V_[j + 1]);

		cusp::blas::axpy(V_[i], V_[j + 1], -H_(i,j));
	}

		H_(j+1,j) = cusp::blas::nrm2(V_[j + 1]);

		if(H_(j+1,j) < 1e-10) break;

		cusp::blas::scal(V_[j + 1], float(1) / H_(j+1,j));

     }
     toc("iteration");


     // scale V_ with beta, i.e. beta*V_, used later for computing simulation trace
     tic();
     for(int i = 0; i < maxiter; i++)
     {
        cusp::blas::scal(V_[i],beta);
     }
     toc("scaling matrix V with beta");
     

     // get matrix H (m x m dimension)
     tic(); 
     for(int rowH=0;rowH < maxiter; rowH++)
     for(int colH = 0; colH <maxiter; colH++)
		H(rowH,colH) = H_(rowH,colH);
     toc("get matrix H -- (m x m) dimension");


     // copying H matrix to np.ndarray
     tic();
    
     for (int i = 0; i < numIter; ++i )
	    for (int k = 0; k < numIter; ++k)
		    result_H[i*numIter + k] = H_(i,k);       
     toc("copying H to np.ndarray");
     
     if(j < maxiter)
     return j+1;
     else return maxiter;
}

int _arnoldi_initVectorPos(int basic_initVector_pos, double* result_H, int size, int numIter)
{   
    if (curMatrix == 0)
        error("loadMatrix must be called before running arnoldi algorithm");
    
    // create initial basic vector on device memory
    tic();

    cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> deviceInitVec(size,0);

    deviceInitVec[basic_initVector_pos] = 1;
    toc("create initial basic vector on device memory based on its position, i.e. basic_initVector_pos");
    
    // system dimension 
    tic();
    int N = size;
    toc("get system dimension");

    // maximum number of Iteration of Arnoldi algorithm
    tic();
    int maxiter = std::min(N, numIter);
    toc("get maximum number of iteration of arnoldi algorithm");

    // create matrix H_ in device memory for iteration
    tic();	
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H_(maxiter + 1, maxiter, 0);
    toc("create matrix H_ in device memory for iteration");

    // returned matrix H after iteration -- Hm in the algorithm -- (m x m) matrix 
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H(maxiter, maxiter); 
    toc("create returned matrix H after iteration -- Hm in the algorithm -- (m x m) matrix ");

    // create matrix V_ for iteration
    tic();
    V_.resize(maxiter+1);
    for (int i = 0; i < maxiter + 1; i++)
        V_[i].resize(N);
    toc("create matrix V_ for iteration");

    // copy initial vector into V_[0]
    tic(); 
    cusp::copy(deviceInitVec,V_[0]); 
    toc("copy initial vector into V_[0]"); 

    // iteration
    tic();
    int j;
    printf("running Arnoldi algorithm...");
    for(j = 0; j < maxiter; j++)
    {
        
	cusp::multiply(*curMatrix, V_[j], V_[j + 1]);
     
	for(int i = 0; i <= j; i++)
	{
		H_(i,j) = cusp::blas::dot(V_[i], V_[j + 1]);

		cusp::blas::axpy(V_[i], V_[j + 1], -H_(i,j));
	}

		H_(j+1,j) = cusp::blas::nrm2(V_[j + 1]);

		if(H_(j+1,j) < 1e-10)  break;     

		cusp::blas::scal(V_[j + 1], float(1) / H_(j+1,j));

     }
     toc("iteration time of Arnoldi algorithm");
     
         
     // get matrix H (m x m dimension)
     tic(); 
     for(int rowH=0;rowH < maxiter; rowH++)
	    for(int colH = 0; colH <maxiter; colH++)
		    H(rowH,colH) = H_(rowH,colH);
     toc("get matrix H -- (m x m) dimension");


     // copying H matrix to np.ndarray
     tic();
    
     for (int i = 0; i < numIter; ++i )
	    for (int k = 0; k < numIter; ++k)
		    result_H[i*numIter + k] = H_(i,k);       
     toc("copying H to np.ndarray");

     if(j < maxiter)
     return j+1;
     else return maxiter;
}


int _arnoldi_parallel(int size, int numIter,double* result_H)
{   
    if (curMatrix == 0)
        error("loadMatrix must be called before running arnoldi algorithm");
    
    // maximum number of Iteration of Arnoldi algorithm
    tic();
    int maxiter = std::min(size, numIter);
    toc("get maximum number of iteration of arnoldi algorithm");

    // create matrix V_all to contain all matrix V: V_all = [V0 V1 ...Vm]
    // V0 = [V0_1 ... V0_n] is (n x n) matrix containing all initial vectors of n-dimensions system
    // Vi = [Vi_1 ... Vi_n] is (n x n) matrix containing all i-th vectors in step i of Arnoldi algorithm
    
    tic();
    V_all.resize(maxiter+1);
    toc("create matrix V_all to contain all matrix Vm");

    // create matrix V_all_final to contain all matrix V; V_all_final = [Vm_0 Vm_2 ...Vm_(n-1)]
    // Vm_0 is the matrix (n x m) V (obtained from Arnoldi algorithm) that corresponds to the 0-th initial vector  
    // Vm_i is the (n x m) matrix V (obtained from Arnoldi algorithm) that corresponds to the i-th initial vector
    
    tic();
    V_all_final.resize(size);
    toc("create matrix V_all_final to contain all matrix Vm");    

    // create matrix H_all to contain all matrix H: H_all = [Hm_1 Hm_2 ...Hm_n]
    // Hm_1, Hm_2 , ... Hm_n are m x m matrices, Hm_i is conresponding to the initial vector i 
        
    tic();
    H_all.resize(size+1);
    toc("create matrix H_all to contain all matrix H");

     // create initial basic vector V_all[0] = n-dimension identity mat
    tic();
 
    cusp::array2d<FLOAT_TYPE, MEMORY_TYPE> Hmat_k(maxiter+1,maxiter,0);
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> Imat(size,size,0);
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE,cusp::column_major> Vm(size,maxiter,0);
    
    for (int i = 0; i< size; i++){
        cusp::copy(Hmat_k,H_all[i]); // initialize H_all[i]
        cusp::copy(Vm,V_all_final[i]); // initialize V_finall_all[k]
    }
    for (int i = 0; i < maxiter+1; i++)
        cusp::copy(Imat,V_all[i]);

    for (int i = 0; i < size; i++)
        Imat(i,i) = 1;

    cusp::copy(Imat,V_all[0]);
    
    toc("create all initial basic vector on device memory V_all[0]");
    
    // Arnoldi parallel algorithm iteration
    
    tic();
    int j;
    int mem = 0; // memorize where break condition happens

    printf("running Arnoldi Algorithm in parallel ...\n");
    
    for (j = 0; j < maxiter; j++){

        cusp::multiply(*curMatrix,V_all[j],V_all[j+1]);
  
        for(int k = 0; k < size; k++){
            // compute Hm-k
          
            cusp::copy(H_all[k],Hmat_k); // Load k-th Hmat matrix         
                
            for(int i = 0; i <= j; i++){
                
                Hmat_k(i,j) = cusp::blas::dot(V_all[i].column(k), V_all[j+1].column(k));
                
                cusp::blas::axpy(V_all[i].column(k), V_all[j+1].column(k), -Hmat_k(i,j));

            }
            
            Hmat_k(j+1,j) = cusp::blas::nrm2(V_all[j+1].column(k));

		    if(Hmat_k(j+1,j) < 1e-10) {

                // an interesting problem: given a system matrix A, different initial vector can produce
                // different number of iteration, i.e, the actual number of iteration
                // To do arnoldi in parallel, we neglect the break condition as in the function arnoldi_initVectorPos
                // We make all vector has the same number of iteration and equal to maxiter
                // i.e., actual_numIter = maxiter (user input parameter)

               
                if (mem == 1){
                    ;
                }
                else {
                    printf("***Notice***: break condition of Arnoldi algorithm is neglected for the initial vector V_%d \n", k);
                    printf("***Notice***: the actual number of iteration corresponding to initial vector V_%d is %d \n", k, j+1);
                    mem = 1;
                }
                
                Hmat_k(j+1,j) = 0;
                cusp::blas::scal(V_all[j+1].column(k),float(0));
                //break;
            }
            else  cusp::blas::scal(V_all[j+1].column(k), float(1) / Hmat_k(j+1,j));           

            cusp::copy(Hmat_k,H_all[k]); // update the k-th Hmatrix           
            
        }
  
    }
    
    toc("iteration time of parallel Arnoldi algorithm");   
    
      // copying H matrix to np.ndarray
     tic();
     cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H(maxiter+1,maxiter,0);
     for (int k = 0; k< size; ++k){
         cusp::copy(H_all[k],H);
         for (int i = 0; i < maxiter; ++i){
             for(int l = 0; l < maxiter; ++l)
                 result_H[i*maxiter + l + k*maxiter*maxiter] = H(i,l);
         }
     }
          
     toc("copying H matrix to np.ndarray");
    
     // save all matrix Vm into V_all_final
     tic();
     for (int k = 0; k < size; k++)     
         for(int i = 0; i < maxiter; i++)
             cusp::blas::copy(V_all[i].column(k),V_all_final[k].column(i));
     
     toc("save all matrix Vm into V_all_final");
     

    return maxiter;
    
}


int _arnoldi_parallel2(int size, int numIter,double* result_H)
{   
    if (curMatrix == 0)
        error("loadMatrix must be called before running arnoldi algorithm");
    
    // maximum number of Iteration of Arnoldi algorithm
    tic();
    int maxiter = std::min(size, numIter);
    toc("get maximum number of iteration of arnoldi algorithm");
    
    tic();
    V_all_final.resize(size);
    toc("create matrix V_all_final to contain all matrix Vm");          

    // create matrix H_ in device memory for iteration
    tic();	
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> H_(maxiter + 1, maxiter, 0);
    toc("create matrix H_ in device memory for iteration");

    cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> deviceInitVec(size,0);
    cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> zeroVec(size,0);
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE,cusp::column_major> V(size,maxiter+1,0); // for iteration
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE,cusp::column_major> Vm(size,maxiter,0); // result
  
    // arnoldi algorithm iteration

    tic();

    for (int l = 0; l < size;  l++){
        cusp::blas::copy(zeroVec,deviceInitVec);
        deviceInitVec[l] = 1;
        cusp::blas::copy(deviceInitVec,V.column(0));//initial vector
        

        int j;
        for(j = 0; j < maxiter; j++){
        
            cusp::multiply(*curMatrix, V.column(j), V.column(j+1));
    
	        for(int i = 0; i <= j; i++){
	    
                H_(i,j) = cusp::blas::dot(V.column(j), V.column(j + 1));

                cusp::blas::axpy(V.column(j), V.column(j+1), -H_(i,j));
	        }

            H_(j+1,j) = cusp::blas::nrm2(V.column(j+1));

		    if(H_(j+1,j) < 1e-10){
                H_(j+1,j) = 0;
                cusp::blas::scal(V.column(j + 1), float(0));
                break;     
            }
            else  cusp::blas::scal(V.column(j+1), float(1) / H_(j+1,j));

            cusp::blas::copy(V.column(j),Vm.column(j));
            
        }
        
        cusp::copy(Vm,V_all_final[l]); // save the matrix Vm for computing simulation result
        
       // copy Hm to result_H 
       for (int i = 0; i < maxiter; i++)
            for(int k = 0; k < maxiter; k++)
                 result_H[i*maxiter + k + l*maxiter*maxiter] = H_(i,k);
    
    }
    
    toc("Arnoldi algorithm iteration");

    return maxiter;

    
}


void _sim(double* matrix_Hf, double* sim_result, int size, int actual_numIter, int numStep)
{
    // compute the simulation result and copy the result back to the CPU (in the sim_result variable)  	  
    // copy matrix Hf to device memory
    numStepOfSim = numStep;
    systemSize   = size;
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> deviceMatrix_Hf(actual_numIter,numStep);
    for(int k=0; k< numStep ; k++)
       for(int i = 0; i < actual_numIter; i++)
          deviceMatrix_Hf(i,k) = matrix_Hf[i*numStep+k];
    toc("copying matrix Hf to device memory");

    // compute simulation result 
    tic();
    device_sim_result.resize(numStep);
    for (int i = 0; i < numStep; i++)
        device_sim_result[i].resize(size);        
    toc("create simulation result matrix");

    tic();
    for (int i=0; i< numStep; i++)
    {
        for(int k=0; k < actual_numIter; k++)
        {
	      cusp::blas::axpy(V_[k], device_sim_result[i], deviceMatrix_Hf(k,i));
        }

    }
    toc("compute simulation result");

    // copy simulation result to np.ndarray
    tic();
    for(int i = 0; i < numStep; i++)
       for(int k = 0; k < size; k++)
           sim_result[i*numStep + k] = device_sim_result[i][k];		
		
    toc("copy simulation result to np.ndarray");
           
}

void _sim2(double* matrix_Hf, int size, int actual_numIter, int numStep)
{
    numStepOfSim = numStep;
    systemSize   = size;
    // compute simulation result and save on the device memory. Do not return the simulation result back to CPU     	  
    // copy matrix Hf to device memory
    tic();
    cusp::array2d<FLOAT_TYPE,MEMORY_TYPE> deviceMatrix_Hf(actual_numIter,numStep);
    for(int k=0; k< numStep ; k++)
       for(int i = 0; i < actual_numIter; i++)
          deviceMatrix_Hf(i,k) = matrix_Hf[i*numStep+k];
    toc("copying matrix Hf to device memory");

    // compute simulation result 
    tic();
    device_sim_result.resize(numStep);
    for (int i = 0; i < numStep; i++)
        device_sim_result[i].resize(size);        
    toc("create simulation result matrix");

    tic();
    for (int i=0; i< numStep; i++)
    {
        for(int k=0; k < actual_numIter; k++)
        {
	      cusp::blas::axpy(V_[k], device_sim_result[i], deviceMatrix_Hf(k,i));
        }

    }
    toc("compute simulation result without copy the result back to CPU"); 
}


void _getKeySimResult(double* keySimResult)
{   // Get the simulation result in a particular dimension, where the dimension is specified by a sparse matrix
    // Steps for using this function:
    // Step1. Load the direction sparse matrix by calling _loadKeyDirMatrix() function
    // Step2. Run the arnoldi algorithm by calling arnoldi_initVector() or arnoldi_initVectorPos()
    // Step3. Compute the matrix Hf = exp(i*timeStep*Hm)
    // Step4. Call _sim2() to compute the simulation result. This function saves the result on device memory and doesnot copy the result to the CPU
    // Step5. Call this function to get the simulation result at this particular dimension

    tic();
    // check consistency and compute key simulation result
    if (numStepOfSim == 0 || systemSize == 0) // check if there is simulation result in device memory
        printf("\n There is no simulation result. Call _sim2() method first");
    else
    {   printf("\n Number of simulation step = %d",numStepOfSim);
        printf("\n Number of Rows of key direction matrix = %d", keyDirMatrix_w);
        if (keyDirMatrix_h != systemSize) // check consistency between the key direction matrix and system dimension
        {
             printf("\n The number of column of key direction matrix is inconsistent with the system dimension");
             toc("check consistency");
        }   
        else
        {   // create key simulation result in device memory
            tic();
            std::vector< cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> > device_keySimResult(numStepOfSim);
            for (int i = 0; i < numStepOfSim; i++)
                device_keySimResult[i].resize(keyDirMatrix_w);
            toc("\n Create keySimResult in device memory");
            
            // compute key simulation result
            tic();
            
            for(int i = 0; i <numStepOfSim; i++)
            {
                cusp::multiply(*keyDirMatrix, device_sim_result[i],device_keySimResult[i]);
            }
            toc("\n Compute key simulation result");

            // copy key direction simulation result to np.array
            tic();
            for(int i = 0; i < numStepOfSim; i++)
                for(int k = 0; k < keyDirMatrix_w; k++)
                   {
		                keySimResult[i*numStepOfSim + k] = device_keySimResult[i][k];		
                   }
            toc("\n Copy key simulation result to np.array");
               
        }
        
    } 
    
}

void _getKeySimResult_parallel(int size, int numIter, double* expHt_e1_tuples, double* keySimResult_tuples)
{
    // get Simulation result in specific direction defined by keyDirMatrix
    // for one initial vector we have:
    // SimResult = V*exp(H*t)*e1, (V,H) are matrices obtained from Arnoldi algorithm
    // KeySimResult = keyDirMatrix*SimResult

    // we get keySimResult for all initial vectors at one time, the result is saved in keySimResult_tuples   

    std::vector< cusp::array1d <FLOAT_TYPE,MEMORY_TYPE> > V_expHt_e1(size); // contain all V*exp(H*t)*e1
     std::vector< cusp::array1d <FLOAT_TYPE, MEMORY_TYPE> > device_keySimResult_tuples(size);
     
     cusp::array1d<FLOAT_TYPE,MEMORY_TYPE > expHt_e1_col_i(numIter);
     
    // Check consitency
    
    if (keyDirMatrix_h != size) // check consistency between the key direction matrix and system dimension
        {
             printf("\n The number of column of key direction matrix is inconsistent with the system dimension");
             toc("check consistency");
        }
    else{
        // compute key Simulation result in parallel
        
        tic();

        for (int i = 0; i < size; i++){
            V_expHt_e1[i].resize(size);
            device_keySimResult_tuples[i].resize(keyDirMatrix_w);

            for(int k = 0; k < numIter; k++){

                expHt_e1_col_i[k] = expHt_e1_tuples[i*numIter + k];

            }

            cusp::multiply(V_all_final[i], expHt_e1_col_i,V_expHt_e1[i]); // compute V*exp(H*t)*e1
            cusp::multiply(*keyDirMatrix,V_expHt_e1[i],device_keySimResult_tuples[i]); // compute keyDirMatrix * V * exp(H*t) * e1           

            // printf("the %d-th key simulation result corresponding to the %d-th initial vector is: \n", i, i );
            // cusp::print(device_keySimResult_tuples[i]);
        }

        toc("Compute keySimResult in parallel");

        // copy keySimulation Result to np.array

        tic();
        
        for (int i = 0; i < size; i++)
            for (int j = 0; j < keyDirMatrix_w; j++)        
                keySimResult_tuples[i*keyDirMatrix_w + j] = device_keySimResult_tuples[i][j];
      }
    
    
}

int _hasGpu()
{
    int rv = 1;
    
    try
    {
        cusp::array1d<FLOAT_TYPE, cusp::host_memory> hostVec(10);
    
        for (int i = 0; i < 10; ++i)
            hostVec[i] = 0;

        cusp::array1d<FLOAT_TYPE,MEMORY_TYPE> deviceVec(hostVec);
    }
    catch(std::exception &e)
    {
        printf("hasGpu() Failed: %s\n", e.what());
        rv = 0;
    }
    
    return rv;
}

extern "C"
{
int hasGpu()
{
    return _hasGpu();
}

void choose_GPU_or_CPU(char* msg)
{
    _choose_GPU_or_CPU(msg);
}

void loadMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{
    _loadMatrix(w, h, nonZeroRows, nonZeroCols, nonZeroEntries, nonZeroCount);
}


void loadKeyDirMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{
    _loadKeyDirMatrix(w, h, nonZeroRows, nonZeroCols, nonZeroEntries, nonZeroCount);
}
    
int arnoldi_initVector(double* init_vector, double* result_H, int size, int numIter)
{
   return _arnoldi_initVector(init_vector, result_H, size, numIter);
   
}

int arnoldi_initVectorPos(int basic_initVector_pos, double* result_H, int size, int numIter)
{
   return _arnoldi_initVectorPos(basic_initVector_pos, result_H, size, numIter);
}

int arnoldi_parallel(int size, int numIter, double* result_H)
{
    return _arnoldi_parallel(size, numIter,result_H);
}


int arnoldi_parallel2(int size, int numIter, double* result_H)
{
    return _arnoldi_parallel2(size, numIter,result_H);
}


void sim(double* matrix_Hf, double* sim_result, int size, int actual_numIter, int numStep)\
{
    _sim(matrix_Hf, sim_result,  size, actual_numIter, numStep);
}
    
void sim2(double* matrix_Hf, int size, int actual_numIter, int numStep)\
{
    _sim2(matrix_Hf, size, actual_numIter, numStep);

}

void getKeySimResult(double* keySimResult)
{
    _getKeySimResult(keySimResult);
}

    
    void getKeySimResult_parallel(int size, int numIter, double* expHt_tuples, double* keySimResult_tuples)
{
    _getKeySimResult_parallel(size, numIter, expHt_tuples, keySimResult_tuples);   
}

}
