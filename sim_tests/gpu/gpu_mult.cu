// Stanley Bak
// GPU Matrix Multiplication interface using Cusp / Cuda
// June 2017

#include <new>
#include <stdio.h>
#include <stdlib.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/multiply.h>

#include <sys/time.h>

typedef double FLOAT_TYPE;
// shared matrix in device memory
static cusp::hyb_matrix<int, FLOAT_TYPE, cusp::device_memory>* curMatrix = 0;

static int nonZeros = 0;

// timing shared variable
static long lastTicUs = 0;

void error(const char* msg)
{
    printf("Fatal Error: %s\n", msg);
    exit(1);
}

void tic()
{
    struct timeval now;
    
    if(gettimeofday( &now, 0))
        error("gettimeofday");
        
    lastTicUs = 1000000 * now.tv_sec + now.tv_usec;
}

// returns the us elaspsed
long toc(const char* label)
{
    struct timeval now;
    
    if(gettimeofday( &now, 0))
        error("gettimeofday");
        
    long nowUs = 1000000 * now.tv_sec + now.tv_usec;
    long dif = nowUs - lastTicUs;
    
    printf("%s: %.4f ms\n", label, dif / 1000.0);
    
    return dif;
}

void _loadMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{
    tic();
    cusp::coo_matrix<int, FLOAT_TYPE, cusp::host_memory> hostMatrix(w, h, nonZeroCount);
        
    printf("loadMatrix() called, estimated size in memory of sparse matrix: %.2f MB (%d nonzeros)\n", 
        nonZeroCount * (8 + 4 + 4) / 1024.0 / 1024.0, nonZeroCount);

    // initialize matrix entries on host
    int index = 0;
 
    for (unsigned int i = 0; i < nonZeroCount; ++i)
    {
        int row = nonZeroRows[i];
        int col = nonZeroCols[i];
        double val = nonZeroEntries[i];
        
        hostMatrix.row_indices[index] = row;
        hostMatrix.column_indices[index] = col;
        hostMatrix.values[index++] = val;
    }
    
    toc("creating host coo matrix");
    
    tic();
    if (curMatrix != 0)
    {
        delete curMatrix;
        curMatrix = 0;
    }
    
    curMatrix = new (std::nothrow) cusp::hyb_matrix<int, FLOAT_TYPE,cusp::device_memory>(hostMatrix);
        
    if (curMatrix == 0)
        error("allocation of heap-based csr matrix in device memory returned nullptr");
        
    toc("copying matrix to device memory");
    nonZeros = nonZeroCount;
}

void _multiply(double* vector, double* result, int size)
{
    if (curMatrix == 0)
        error("loadMatrix must be called before multiply");
    
    // initialize input vector
    tic();
    cusp::array1d<FLOAT_TYPE, cusp::host_memory> hostVec(size);
    
    for (int i = 0; i < size; ++i)
        hostVec[i] = vector[i];
    toc("creating hostVec vector");
    
    tic();
    cusp::array1d<FLOAT_TYPE,cusp::device_memory> deviceVec(hostVec);
    toc("copying vector to device memory");
    
    // create device vec; should be negligible time, don't even measure
    cusp::array1d<FLOAT_TYPE, cusp::device_memory> resultVec(size);
    
    // compute result = A * stat
    tic();
    cusp::multiply(*curMatrix, deviceVec, resultVec);
    hipDeviceSynchronize(); // wait until prior kernel is finished
    long usElapsed = toc("matrix-vector multiplication");
    
    // each nonzero is 2 FLOPS (one for the multiplication, and another for an addition
    // microseconds (us) is 1000 * 1000, which is close to megaflops
    double megaFlopsPerSecond = 2 * nonZeros / usElapsed; 
    printf("achieved megaflops = %f\n", megaFlopsPerSecond);
    
    tic();
    cusp::array1d<FLOAT_TYPE, cusp::host_memory> resultHost(resultVec);
    toc("copying result to host memory");
    
    tic();
    for (int i = 0; i < size; ++i)
        result[i] = resultHost[i];
        
    toc("copying to np.ndarray");
}

int _hasGpu()
{
    int rv = 1;
    
    try
    {
        cusp::array1d<FLOAT_TYPE, cusp::host_memory> hostVec(10);
    
        for (int i = 0; i < 10; ++i)
            hostVec[i] = 0;

        cusp::array1d<FLOAT_TYPE,cusp::device_memory> deviceVec(hostVec);
    }
    catch(std::exception &e)
    {
        printf("hasGpu() Failed: %s\n", e.what());
        rv = 0;
    }
    
    return rv;
}

extern "C"
{
int hasGpu()
{
    return _hasGpu();
}

void loadMatrix(int w, int h, int* nonZeroRows, int* nonZeroCols, double* nonZeroEntries, int nonZeroCount)
{
    _loadMatrix(w, h, nonZeroRows, nonZeroCols, nonZeroEntries, nonZeroCount);
}

void multiply(double* vector, double* result, int size)\
{
    _multiply(vector, result, size);
}

}
